#include "hip/hip_runtime.h"
#include <algorithm>
#include "../inc/RayTracer_Dynamic_AABB.cuh"
#include "../inc/RayTracer_DynamicSkeleton.cuh"

// TODO: test
#include <stdio.h>


// TODO: although this file is called AABB
// but the purpose of this file is for the interface controlling difference ray intersection accleration structure
// therefore, renaming this file is needed
// the detail of each acceleration structure should be moved to its own file
// which mean to create a new file for each acceleration structure


// Define
// ...


// Typedef
typedef int32_t(*compare_func_t)(const void*, const void*);


// Data Structure
struct RecordBounding {
	Bounding				bounding;
	SceneObject_Hitable		*hitable;
};


// Static Function Prototype
// skeleton
Dynamic_CUDA_constructTypeSkeleton(aabb_default, Hitable_AABB, Hitable_AABB);

// host
__host__ static inline void	host_aabb_getBounding		(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size);
__host__ static void		host_aabb_buildTree			(AABB* *dst, RecordBounding *record_list, int32_t size);
__host__ static void		host_aabb_createAABB		(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right);

// global
__global__ static void		global_aabb_getBounding		(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size, int32_t offset);
__global__ static void		global_aabb_createAABB		(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right);
__global__ static void		global_aabb_setRoot			(Hitable_AABB *hitable_aabb, AABB *aabb);

// device
// ...

// compare function
__host__ static inline int32_t	select_compare_func			(RecordBounding *record_list, int32_t size);
__host__ static fp_t			get_diff					(RecordBounding *record_list, int32_t size, int32_t axis);
__host__ static int32_t			compare_x_axis				(const void *a, const void *b);
__host__ static int32_t			compare_y_axis				(const void *a, const void *b);
__host__ static int32_t			compare_z_axis				(const void *a, const void *b);


// Static Datas
static AABB*			*buffer_aabb;
static compare_func_t	compare_func_list[] = {
	compare_x_axis,
	compare_y_axis,
	compare_z_axis
};


// Operation Handling
__host__ void RayTracer_Dynamic_AABB_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// ...

	// create type
	Dynamic_ContainerType *type;
	Dynamic_CUDA_addType(aabb_default,	aabb_default,	type_list);

	// buffer
	hipMalloc(&buffer_aabb, sizeof(AABB*));
}


__host__ void RayTracer_Dynamic_AABB_info() {
}


__host__ void RayTracer_Dynamic_AABB_del() {
	hipFree(buffer_aabb);
}


__host__ error_t Dynamic_AABB_load(Hitable_AABB *hitable_aabb, SceneObject_Hitable* *hitable_list, int32_t size) {
	// get bounding
	RecordBounding *record_list;
	record_list = (RecordBounding*)malloc(size * sizeof(RecordBounding));

	host_aabb_getBounding(record_list, hitable_list, size);

	// build tree
	// TODO: should global_aabb_setRoot be inside host_aabb_buildTree ?
	AABB *aabb_device;
	host_aabb_buildTree(&aabb_device, record_list, size);
	global_aabb_setRoot <<< 1, 1 >>> (hitable_aabb, aabb_device);

	// free space
	free(record_list);

	return ERROR_NO;
}


// Static Function Implementation
// host
__host__ static inline void host_aabb_getBounding(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size) {
	RecordBounding			*record_list_device;
	SceneObject_Hitable*	*hitable_list_device;

	// allocate device space
	hipMalloc(&record_list_device, size * sizeof(RecordBounding));
	hipMalloc(&hitable_list_device, size * sizeof(SceneObject_Hitable*));
	
	// transfer hitable_list from host to device
	hipMemcpy(hitable_list_device, hitable_list, size * sizeof(SceneObject_Hitable*), hipMemcpyHostToDevice);

	// TODO: the number of thread should be dynamic
	global_aabb_getBounding <<< 1, 10 >>> (record_list_device, hitable_list_device, size, 10);

	// transfer record_list from device to host
	hipMemcpy(record_list, record_list_device, size * sizeof(RecordBounding), hipMemcpyDeviceToHost);

	// free device space
	hipFree(record_list_device);
	hipFree(hitable_list_device);
}


__host__ static void host_aabb_buildTree(AABB* *dst, RecordBounding *record_list, int32_t size) {	
	// base case
	// when size <= 2, then this aabb node is a leaf
	if (size == 0) {
		host_aabb_createAABB(dst, nullptr, nullptr, nullptr, nullptr);
		return;
	}

	if (size == 1) {
		host_aabb_createAABB(dst, nullptr, nullptr, record_list[0].hitable, nullptr);
		return;
	}

	if (size == 2) {
		host_aabb_createAABB(dst, nullptr, nullptr, record_list[0].hitable, record_list[1].hitable);
		return;
	}

	// recursive case
	// sort the list
	int32_t index_compare = select_compare_func(record_list, size);
	qsort(record_list, size, sizeof(RecordBounding), compare_func_list[index_compare]);

	// build the tree recursively
	AABB *left, *right;
	host_aabb_buildTree(&left, record_list, size / 2);
	host_aabb_buildTree(&right, record_list + size / 2, size - size / 2);
	
	// build this level of aabb
	host_aabb_createAABB(dst, left, right, nullptr, nullptr);
}


__host__ static void host_aabb_createAABB(
	AABB* *dst, 
	AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right) {

	// create aabb
	global_aabb_createAABB <<< 1, 1 >>> (buffer_aabb, aabb_left, aabb_right, hitable_left, hitable_right);

	// transfer this pointer from device to host
	hipMemcpy(dst, buffer_aabb, sizeof(AABB*), hipMemcpyDeviceToHost);
}


// global
__global__ static void	global_aabb_getBounding(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size, int32_t offset) {
	// parallle processing
	int32_t global_index = blockIdx.x * blockDim.x + threadIdx.x; 

	// just copying (copy constructor)
	for (int32_t i = 0; i < size; i += offset) {
		if (i + global_index >= size) continue;

		record_list[i + global_index].bounding		= hitable_list[i + global_index]->bounding;
		record_list[i + global_index].hitable		= hitable_list[i + global_index];
	}
}


__global__ static void	global_aabb_createAABB(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right) {
	*dst = new AABB(aabb_left, aabb_right, hitable_left, hitable_right);
}


__global__ static void	global_aabb_setRoot(Hitable_AABB *hitable_aabb, AABB *aabb) {
	hitable_aabb->setAABB(aabb);
}


// device
// ...


// compare function
__host__ static inline int32_t select_compare_func(RecordBounding *record_list, int32_t size) {
	/* The purpose of this function is to find out in which axis
	   the sorting will result the best / better for tree building

	   Currently, it is achieved by find the diff between min and max of the value of
	   midpoint of bounding box in each axis
	   The larger the differences for the axis
	   the larger the possibility that the sort in this axis will be better
	 */

	 // find the difference of min and max
	 fp_t diff_x = get_diff(record_list, size, 0);
	 fp_t diff_y = get_diff(record_list, size, 1);
	 fp_t diff_z = get_diff(record_list, size, 2);

	 // return the axis index
	 return diff_x > diff_y && diff_x > diff_z ? 0 : diff_y > diff_z ? 1 : 2;
}


__host__ static fp_t get_diff(RecordBounding *record_list, int32_t size, int32_t axis) {
	if (size == 0) return 0;

	fp_t min_, max_, mid;
	mid = 
		record_list[0].bounding.min_[axis] + 
		(record_list[0].bounding.max_[axis] - record_list[0].bounding.min_[axis]) / 2;
	
	min_ = mid;
	max_ = mid;

	for (int32_t i = 1; i < size; ++i) {
		mid = 
			record_list[i].bounding.min_[axis] + 
			(record_list[i].bounding.max_[axis] - record_list[i].bounding.min_[axis]) / 2;

		min_ = std::min<fp_t>(mid, min_);
		max_ = std::max<fp_t>(mid, max_);
	}

	// return std::abs<int32_t>(max_ - min_);
	return max_ - min_;
}


__host__ static int32_t compare_x_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[0] + bounding_1->bounding.max_[0];
	int32_t	mid_2	= bounding_2->bounding.min_[0] + bounding_2->bounding.max_[0];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}


__host__ static int32_t compare_y_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[1] + bounding_1->bounding.max_[1];
	int32_t	mid_2	= bounding_2->bounding.min_[1] + bounding_2->bounding.max_[1];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}


__host__ static int32_t compare_z_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[2] + bounding_1->bounding.max_[2];
	int32_t	mid_2	= bounding_2->bounding.min_[2] + bounding_2->bounding.max_[2];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}

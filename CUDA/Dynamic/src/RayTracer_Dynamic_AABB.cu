#include "hip/hip_runtime.h"
#include <algorithm>
#include "../inc/RayTracer_Dynamic_AABB.cuh"
#include "../inc/RayTracer_DynamicSkeleton.cuh"


// Define
// ...


// Typedef
// ...


// Data Structure
// ...


// Static Function Prototype
// skeleton
Dynamic_CUDA_constructTypeSkeleton(aabb_default, Hitable_AABB, Hitable_AABB);

// cuda linker function
// TODO: remove
/*
__global__ static void aabb_load	(SceneObject_AABB *aabb, SceneObject_Hitable *hitable_list, int32_t size);
*/

// TODO: need to be standardized (although these function are static)
__global__ static void	aabb_getBounding	(Bounding *bounding_list, SceneObject_Hitable *hitable_list, int32_t size);
__global__ static void	aabb_createAABB		(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right);
__global__ static void	aabb_setRoot		(Hitable_AABB *aabb, AABB *root);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_AABB_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// ...

	// create type
	Dynamic_ContainerType *type;
	Dynamic_CUDA_addType(aabb_default,	aabb_default,	type_list);
}


__host__ void RayTracer_Dynamic_AABB_info() {
}


__host__ void RayTracer_Dynamic_AABB_del() {
}


__host__ error_t Dynamic_AABB_load(Hitable_AABB *aabb, SceneObject_Hitable* *hitable_list, int32_t size) {
	// TODO: remove
	/*
	// data to device
	SceneObject_Hitable *hitable_list_device;
	hipMalloc(&hitable_list_device, size * sizeof(SceneObject_Hitable*));
	hipMemcpy(hitable_list_device, hitable_list, size * sizeof(SceneObject_Hitable*));

	// device function
	aabb_load <<< 1, 1 >>> (aabb, hitable_list_device, size);

	// free space
	hipFree(hitable_device);
	
	return ERROR_NO;
	*/

	// TODO: not yet completed
	// get bounding

	// build tree

	return ERROR_NO;
}


// Static Function Implementation
// cuda linker function
// TODO: remove
/*
__global__ static void aabb_load(SceneObject_AABB *aabb, SceneObject_Hitable *hitable_list, int32_t size) {
	Hitable_AABB *aabb = Hitable_AABB::create(hitable_list, size, 1);
	if (aabb == nullptr) return;

	aabb->addHitable(aabb)
}
*/


__global__ static void	aabb_getBounding(Bounding *bounding_list, SceneObject_Hitable *hitable_list, int32_t size) {
	// just copying (copy constructor)
	for (int32_t i = 0; i < size; ++i) {
		bounding_list[i] = hitable_list[i].bounding;
	}
}


__global__ static void	aabb_createAABB(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right) {
	*dst = new AABB(aabb_left, aabb_right, hitable_left, hitable_right);
}


__global__ static void	aabb_setRoot(Hitable_AABB *aabb, AABB *root) {
	aabb->setAABB(root);
}

#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Dynamic_Base.cuh"
#include <stdio.h>


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
__global__ static void global_Base_init	(int32_t size_random);


// Operation Handling
__host__ void RayTracer_Dynamic_Base_init() {
	global_Base_init <<< 1, 1 >>> (100);
}


// Static Function Implementation
__global__ static void global_Base_init(int32_t size_random) {
	UtilMath::initUtilMath(size_random);
}

#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Camera.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototpye
// skeleton
Dynamic_CUDA_constructTypeSkeleton(camera_default,	Camera,	Camera);

// table
// ...

// cuda linker function
// __global__ static void	camera_setAll			(Camera *camera);
__global__ static void	camera_setLookFrom		(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2);
__global__ static void	camera_setLookAt		(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2);
__global__ static void	camera_setUpDirection	(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2);
__global__ static void	camera_setFOV			(Camera *camera, fp_t value);
__global__ static void	camera_setAspectRatio	(Camera *camera, fp_t value);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Camera_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// ...

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(camera_default,	camera_default,	type_list);
}


__host__ void RayTracer_Dynamic_Camera_info() {
}


__host__ void RayTracer_Dynamic_Camera_del() {
}


// cuda linker function
// __host__ void Dynamic_Camera_setAll(Camera *camera) {
// }


__host__ error_t Dynamic_Camera_setLookFrom(Camera *camera, const Vec3f look_from) {
	camera_setLookFrom <<< 1, 1 >>> (camera, look_from[0], look_from[1], look_from[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Camera_setLookAt(Camera *camera, const Vec3f look_at) {
	camera_setLookAt <<< 1, 1 >>> (camera, look_at[0], look_at[1], look_at[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Camera_setUpDirection(Camera *camera, const Vec3f up) {
	camera_setUpDirection <<< 1, 1 >>> (camera, up[0], up[1], up[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Camera_setFOV(Camera *camera, fp_t value) {
	camera_setFOV <<< 1, 1 >>> (camera, value);
	return ERROR_NO;
}


__host__ error_t Dynamic_Camera_setAspectRatio(Camera *camera, fp_t value) {
	camera_setAspectRatio <<< 1, 1 >>> (camera, value);
	return ERROR_NO;
}


// Static Function Implementation
// table
// ...

// cuda linker function
// __global__ static void camera_setAll(Camera *camera) {
// }


__global__ static void camera_setLookFrom(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2) {
	camera->setLookFrom(Vec3f(v_0, v_1, v_2));
}


__global__ static void camera_setLookAt(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2) {
	camera->setLookAt(Vec3f(v_0, v_1, v_2));
}


__global__ static void camera_setUpDirection(Camera *camera, fp_t v_0, fp_t v_1, fp_t v_2) {
	camera->setUpDirection(Vec3f(v_0, v_1, v_2));
}


__global__ static void camera_setFOV(Camera *camera, fp_t value) {
	camera->setFOV(value);
}


__global__ static void camera_setAspectRatio(Camera *camera, fp_t value) {
	camera->setAspectRatio(value);
}

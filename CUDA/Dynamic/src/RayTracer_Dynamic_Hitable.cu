#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Hitable.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
__global__ static void			config_sphere_setCenter				(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_sphere_setRadius				(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_trimesh_setPoint_0			(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_trimesh_setPoint_1			(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_trimesh_setPoint_2			(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// __global__ static void			interact_aabb_addHitable			(int *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_aabb_rmHitable				(int *ret, void *object, void* *list, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(sphere,	SceneObject_Hitable,	Hitable_Sphere);
Dynamic_CUDA_constructTypeSkeleton(trimesh,	SceneObject_Hitable,	Hitable_Trimesh);
// Dynamic_CUDA_constructTypeSkeleton(aabb,		SceneObject_Hitable,	Hitable_AABB);

Dynamic_CUDA_constructTypeConfigLinker(sphere_setCenter,		config_sphere_setCenter);
Dynamic_CUDA_constructTypeConfigLinker(sphere_setRadius,		config_sphere_setRadius);
Dynamic_CUDA_constructTypeConfigLinker(trimesh_setPoint_0,	config_trimesh_setPoint_0);
Dynamic_CUDA_constructTypeConfigLinker(trimesh_setPoint_1,	config_trimesh_setPoint_1);
Dynamic_CUDA_constructTypeConfigLinker(trimesh_setPoint_2,	config_trimesh_setPoint_2);

// Dynamic_CUDA_constructTypeInteractLinker(aabb_addHitable,	interact_aabb_addHitable);
// Dynamic_CUDA_constructTypeInteractLinker(aabb_rmHitable,		interact_aabb_rmHitable);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Hitable_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	Dynamic_CUDA_addTypeConfigLinker(sphere,		sphere_setCenter);
	Dynamic_CUDA_addTypeConfigLinker(sphere,		sphere_setRadius);
	Dynamic_CUDA_addTypeConfigLinker(trimesh,	trimesh_setPoint_0);
	Dynamic_CUDA_addTypeConfigLinker(trimesh,	trimesh_setPoint_1);
	Dynamic_CUDA_addTypeConfigLinker(trimesh,	trimesh_setPoint_2);

	// Dynamic_CUDA_addTypeInteractLinker(aabb,		aabb_addHitable);
	// Dynamic_CUDA_addTypeInteractLinker(aabb,		aabb_rmHitable);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(sphere,		sphere,		type_list);
	Dynamic_CUDA_addType(trimesh,	trimesh,	type_list);
	// Dynamic_CUDA_addType(aabb,		aabb,		type_list);
}


__host__ void RayTracer_Dynamic_Hitable_info() {
}


__host__ void RayTracer_Dynamic_Hitable_del() {
}


// Static Function Implementation
// ...


// table
__global__ static void config_sphere_setCenter(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Hitable_Sphere		*hitable	= (Hitable_Sphere*)object;
	double				*center		= (double*)data;

	hitable->setCenter(Vec3f(center[0], center[1], center[2]));
	*ret = 0;
}


__global__ static void config_sphere_setRadius(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Hitable_Sphere		*hitable	= (Hitable_Sphere*)object;
	double				radius		= *((double*)data);

	hitable->setRadius(radius);
	*ret = 0;
}


__global__ static void config_trimesh_setPoint_0(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Hitable_Trimesh		*hitable	= (Hitable_Trimesh*)object;
	double				*point		= (double*)data;
	Vec3f				vec_point	= Vec3f(point[0], point[1], point[2]);

	hitable->setPoint(vec_point, hitable->point[1], hitable->point[2]);
	*ret = 0;
}


__global__ static void config_trimesh_setPoint_1(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Hitable_Trimesh		*hitable = (Hitable_Trimesh*)object;
	double				*point		= (double*)data;
	Vec3f				vec_point	= Vec3f(point[0], point[1], point[2]);

	hitable->setPoint(hitable->point[0], vec_point, hitable->point[2]);
	*ret = 0;
}


__global__ static void config_trimesh_setPoint_2(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Hitable_Trimesh		*hitable = (Hitable_Trimesh*)object;
	double				*point		= (double*)data;
	Vec3f				vec_point	= Vec3f(point[0], point[1], point[2]);

	hitable->setPoint(hitable->point[0], hitable->point[1], vec_point);
	*ret = 0;
}


// __global__ static void interact_aabb_addHitable(int8_t *ret, void *object, void* *list, uint32_t size) {
// 	Hitable_AABB		*hitable	= (Hitable_AABB*)object;
// 	SceneObject_Hitable	*child		= (SceneObject_Hitable*)(list[0]);

// 	if (!hitable->addHitable(child)) return -1;
// 	*ret = 0;
// }


// __global__ static void interact_aabb_rmHitable(int8_t *ret, void *object, void* *list, uint32_t size) {
// 	Hitable_AABB		*hitable	= (Hitable_AABB*)object;
// 	SceneObject_Hitable	*child		= (SceneObject_Hitable*)(list[0]);

// 	if (!hitable->rmHitable(child)) return -1;
// 	*ret = 0;
// }

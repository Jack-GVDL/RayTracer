#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Light.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
__global__ static void			config_directional_setOrientation	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_point_setAttenuation			(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(directional,	SceneObject_Light,	Light_Directional);
Dynamic_CUDA_constructTypeSkeleton(point,		SceneObject_Light,	Light_Point);

Dynamic_CUDA_constructTypeConfigLinker(directional_setOrientation,	config_directional_setOrientation);
Dynamic_CUDA_constructTypeConfigLinker(point_setAttenuation,		config_point_setAttenuation);

// cuda linker function
__global__ static void	light_setOrigin		(SceneObject_Light *light, fp_t v_0, fp_t v_1, fp_t v_2);
__global__ static void	light_setColor		(SceneObject_Light *light, fp_t v_0, fp_t v_1, fp_t v_2);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Light_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	Dynamic_CUDA_addTypeConfigLinker(directional,	directional_setOrientation);
	Dynamic_CUDA_addTypeConfigLinker(point,			point_setAttenuation);

	// table_config_directional.push_back(	config_directional_setOrientation	);
	// table_config_point.push_back(		config_point_setAttenuation			);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(directional,	directional,	type_list);
	Dynamic_CUDA_addType(point,			point,			type_list);
}


__host__ void RayTracer_Dynamic_Light_info() {
}


__host__ void RayTracer_Dynamic_Light_del() {
}


__host__ error_t Dynamic_Light_setOrigin(SceneObject_Light *light, const Vec3f &origin) {
	light_setOrigin <<< 1, 1 >>> (light, origin[0], origin[1], origin[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Light_setColor(SceneObject_Light *light, const Vec3f &color) {
	light_setColor <<< 1, 1 >>> (light, color[0], color[1], color[2]);
	return ERROR_NO;
}


// Static Function Implementation
// ...


// table
__global__ static void config_directional_setOrientation(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Light_Directional	*light			= (Light_Directional*)object;
	double				*orientation	= (double*)data;
	
	light->setOrientation(Vec3f(orientation[0], orientation[1], orientation[2]));
	*ret = 0;
}


__global__ static void config_point_setAttenuation(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Light_Point			*light	= (Light_Point*)object;
	double				*coeff	= (double*)data;

	light->setAttenuation(Vec3f(coeff[0], coeff[1], coeff[2]));
	*ret = 0;
}


// cuda linker function
__global__ static void light_setOrigin(SceneObject_Light *light, fp_t v_0, fp_t v_1, fp_t v_2) {
	light->setOrigin(Vec3f(v_0, v_1, v_2));
}


__global__ static void light_setColor(SceneObject_Light *light, fp_t v_0, fp_t v_1, fp_t v_2) {
	light->setColor(Vec3f(v_0, v_1, v_2));
}

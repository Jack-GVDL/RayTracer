#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Light.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
__global__ static void			config_directional_setOrientation	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_point_setAttenuation			(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(directional,	SceneObject_Light,	Light_Directional);
Dynamic_CUDA_constructTypeSkeleton(point,		SceneObject_Light,	Light_Point);

Dynamic_CUDA_constructTypeConfigLinker(directional_setOrientation,	config_directional_setOrientation);
Dynamic_CUDA_constructTypeConfigLinker(point_setAttenuation,		config_point_setAttenuation);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Light_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	Dynamic_CUDA_addTypeConfigLinker(directional,	directional_setOrientation);
	Dynamic_CUDA_addTypeConfigLinker(point,			point_setAttenuation);

	// table_config_directional.push_back(	config_directional_setOrientation	);
	// table_config_point.push_back(		config_point_setAttenuation			);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(directional,	directional,	type_list);
	Dynamic_CUDA_addType(point,			point,			type_list);
}


__host__ void RayTracer_Dynamic_Light_info() {
}


__host__ void RayTracer_Dynamic_Light_del() {
}


// Static Function Implementation
// ...


// table
__global__ static void config_directional_setOrientation(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Light_Directional	*light			= (Light_Directional*)object;
	double				*orientation	= (double*)data;
	
	light->setOrientation(Vec3f(orientation[0], orientation[1], orientation[2]));
	*ret = 0;
}


__global__ static void config_point_setAttenuation(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Light_Point			*light	= (Light_Point*)object;
	double				*coeff	= (double*)data;

	light->setAttenuation(Vec3f(coeff[0], coeff[1], coeff[2]));
	*ret = 0;
}

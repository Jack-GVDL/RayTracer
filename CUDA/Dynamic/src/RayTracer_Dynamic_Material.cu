#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Material.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// ...


// Static Data
// skeleton
Dynamic_CUDA_constructTypeSkeleton(material_default,	Material,	Material);

// cuda linker function
__global__ static void	material_addScatter			(Material *material, Scatter *scatter);
__global__ static void	material_rmScatter			(Material *material, Scatter *scatter);
__global__ static void	material_setTransmissive	(Material *material, fp_t v_0, fp_t v_1, fp_t v_2);
__global__ static void	material_setIndex 			(Material *material, fp_t value);


// Operation Handling
__host__ void RayTracer_Dynamic_Material_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// ...

	// create type
	Dynamic_ContainerType *type;
	
	Dynamic_CUDA_addType(material_default,	material_default,	type_list);
}


__host__ void RayTracer_Dynamic_Material_info() {
}


__host__ void RayTracer_Dynamic_Material_del() {
}


__host__ error_t Dynamic_Material_addScatter(Material *material, Scatter *scatter) {
	material_addScatter <<< 1, 1 >>> (material, scatter);
	return ERROR_NO;
}


__host__ error_t Dynamic_Material_rmScatter(Material *material, Scatter *scatter) {
	material_rmScatter <<< 1, 1 >>> (material, scatter);
	return ERROR_NO;
}


__host__ error_t Dynamic_Material_setTransmissive(Material *material, const Vec3f &transmissive) {
	material_setTransmissive <<< 1, 1 >>> (material, transmissive[0], transmissive[1], transmissive[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Material_setIndex(Material *material, fp_t value) {
	material_setIndex <<< 1, 1 >>> (material, value);
	return ERROR_NO;
}


// Static Function Implementation
// cuda linker function
__global__ static void material_addScatter(Material *material, Scatter *scatter) {
	material->addScatter(scatter);
}


__global__ static void material_rmScatter(Material *material, Scatter *scatter) {
	material->rmScatter(scatter);
}


__global__ static void material_setTransmissive(Material *material, fp_t v_0, fp_t v_1, fp_t v_2) {
	material->transmissive->setPixel(
		Vec3f(), Vec3f(v_0, v_1, v_2));
}


__global__ static void material_setIndex(Material *material, fp_t value) {
	material->index = value;
}

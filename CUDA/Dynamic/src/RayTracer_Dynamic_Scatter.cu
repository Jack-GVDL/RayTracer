#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Dynamic_Scatter.cuh"
#include "../inc/RayTracer_DynamicSkeleton.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
__global__ static void			config_random_setRadius				(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_random_setParallel			(int8_t *ret, void *object, uint8_t *data, uint32_t size);
__global__ static void			config_random_setRaySize			(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(light,		Scatter,	Scatter_Light);
Dynamic_CUDA_constructTypeSkeleton(anyHit,		Scatter,	Scatter_AnyHit);
Dynamic_CUDA_constructTypeSkeleton(reflection,	Scatter,	Scatter_Reflection);
Dynamic_CUDA_constructTypeSkeleton(refraction,	Scatter,	Scatter_Refraction);
Dynamic_CUDA_constructTypeSkeleton(gradient,	Scatter,	Scatter_Gradient);
Dynamic_CUDA_constructTypeSkeleton(emitter,		Scatter,	Scatter_Emitter);
Dynamic_CUDA_constructTypeSkeleton(random,		Scatter,	Scatter_Random);

Dynamic_CUDA_constructTypeConfigLinker(random_setRadius,	config_random_setRadius);
Dynamic_CUDA_constructTypeConfigLinker(random_setParallel,	config_random_setParallel);
Dynamic_CUDA_constructTypeConfigLinker(random_setRaySize,	config_random_setRaySize);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Scatter_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	Dynamic_CUDA_addTypeConfigLinker(random,		random_setRadius);
	Dynamic_CUDA_addTypeConfigLinker(random,		random_setParallel);
	Dynamic_CUDA_addTypeConfigLinker(random,		random_setRaySize);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(light,		light,		type_list);
	Dynamic_CUDA_addType(reflection,	reflection,	type_list);
	Dynamic_CUDA_addType(refraction,	refraction,	type_list);
	Dynamic_CUDA_addType(gradient,	gradient,	type_list);
	Dynamic_CUDA_addType(emitter,	emitter,	type_list);
	Dynamic_CUDA_addType(random,		random,		type_list);
	Dynamic_CUDA_addType(anyHit,		any_hit,	type_list);
}


__host__ void RayTracer_Dynamic_Scatter_info() {
}


__host__ void RayTracer_Dynamic_Scatter_del() {
}


// Static Function Implementation
// table
__global__ static void config_random_setRadius(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Scatter_Random	*scatter	= (Scatter_Random*)object;
	double			radius		= ((double*)data)[0];

	scatter->setRadius((fp_t)radius);
	*ret = 0;
}


__global__ static void config_random_setParallel(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Scatter_Random	*scatter	= (Scatter_Random*)object;
	int				is_parallel	= ((int*)data)[0];

	scatter->setParallel(is_parallel ? 1 : 0);
	*ret = 0;
}


__global__ static void config_random_setRaySize(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Scatter_Random	*scatter	= (Scatter_Random*)object;
	int				count		= ((int*)data)[0];

	scatter->setRaySize((uint8_t)UtilMath::clamp<int>(0, count, 255));
	*ret = 0;
}

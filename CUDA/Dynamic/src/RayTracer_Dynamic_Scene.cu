#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Dynamic_Scene.cuh"

// TODO: test
#include <stdio.h>


// Define
// ...


// Typedef
// ...


// Static Data
extern __device__ Scene			*scene;


// Static Function Prototype
__global__ static void		global_addLight			(SceneObject_Light *light);
__global__ static void		global_addHitable		(SceneObject_Hitable *hitable);
__global__ static void		global_rmLight			(SceneObject_Light *light);
__global__ static void		global_rmHitable		(SceneObject_Hitable *hitable);
__global__ static void		global_getLightIndex	(int32_t *dst);
__global__ static void		global_getHitableIndex	(int32_t *dst);
__global__ static void		global_getLightSize		(int32_t *dst);


// Operation Handling
__host__ error_t Dynamic_Scene_addLight(SceneObject_Light *light) {
	global_addLight <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_addHitable(SceneObject_Hitable *hitable) {
	global_addHitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_rmLight(SceneObject_Light * light) {
	global_rmLight <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_rmHitable(SceneObject_Hitable *hitable) {
	global_rmHitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


__host__ int32_t Dynamic_Scene_getLightIndex() {
	int32_t *dst_device;
	int32_t	dst_host = -1;
	hipMalloc(&dst_device, sizeof(int32_t));

	global_getLightIndex <<< 1, 1 >>> (dst_device);

	hipMemcpy(&dst_host, dst_device, sizeof(int32_t), hipMemcpyDeviceToHost);
	hipFree(dst_device);

	return dst_host;
}


__host__ int32_t Dynamic_Scene_getHitableIndex() {
	int32_t *dst_device;
	int32_t	dst_host = -1;
	hipMalloc(&dst_device, sizeof(int32_t));

	global_getHitableIndex <<< 1, 1 >>> (dst_device);

	hipMemcpy(&dst_host, dst_device, sizeof(int32_t), hipMemcpyDeviceToHost);
	hipFree(dst_device);

	return dst_host;
}


__host__ int32_t Dynamic_Scene_getLightSize() {
	int32_t *dst_device;
	int32_t	dst_host = -1;
	hipMalloc(&dst_device, sizeof(int32_t));

	global_getLightSize <<< 1, 1 >>> (dst_device);

	hipMemcpy(&dst_host, dst_device, sizeof(int32_t), hipMemcpyDeviceToHost);
	hipFree(dst_device);

	return dst_host;
}


// Static Function Implementation
__global__ static void global_addLight(SceneObject_Light *light) {
	scene->addLight(light);
}


__global__ static void global_addHitable(SceneObject_Hitable *hitable) {
	scene->addHitable(hitable);
}


__global__ static void global_rmLight(SceneObject_Light *light) {
	scene->rmLight(light);
}


__global__ static void global_rmHitable(SceneObject_Hitable *hitable) {
	scene->rmHitable(hitable);
}


__global__ static void global_getLightIndex(int32_t *dst)  {
	*dst = scene->light_index;
	dst[0] = 11;
}


__global__ static void global_getHitableIndex(int32_t *dst)  {
	*dst = scene->hitable_index;
	*dst = 12;
}


__global__ static void global_getLightSize(int32_t *dst)  {
	// *dst = scene->light_size;
	*dst = 10;
}

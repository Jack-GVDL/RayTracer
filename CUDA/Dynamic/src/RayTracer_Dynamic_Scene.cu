#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Dynamic_Scene.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
extern __device__ Scene			*scene;


// Static Function Prototype
__global__ static void		global_addLight		(SceneObject_Light *light);
__global__ static void		global_addHitable	(SceneObject_Hitable *hitable);
__global__ static void		global_rmLight		(SceneObject_Light *light);
__global__ static void		global_rmHitable	(SceneObject_Hitable *hitable);


// Operation Handling
__host__ error_t Dynamic_Scene_addLight(SceneObject_Light *light) {
	global_addLight <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_addHitable(SceneObject_Hitable *hitable) {
	global_addHitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_rmLight(SceneObject_Light * light) {
	global_rmLight <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ error_t Dynamic_Scene_rmHitable(SceneObject_Hitable *hitable) {
	global_rmHitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


// Static Function Implementation
__global__ static void global_addLight(SceneObject_Light *light) {
	scene->addLight(light);
}


__global__ static void global_addHitable(SceneObject_Hitable *hitable) {
	scene->addHitable(hitable);
}


__global__ static void global_rmLight(SceneObject_Light *light) {
	scene->rmLight(light);
}


__global__ static void global_rmHitable(SceneObject_Hitable *hitable) {
	scene->rmHitable(hitable);
}

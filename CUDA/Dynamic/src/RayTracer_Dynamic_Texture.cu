#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Texture.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
// __global__ static void			config_convolutor_setKernelSize		(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_convolutor_setKernelArray	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_checkerboard_setBoardSize	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_additor_setAdditor			(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_multiplier_setMultiplier		(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// __global__ static void			interact_convolutor_setTexture		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_mapSphere_setSphere		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_dirSphere_setSphere		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_mapTrimesh_setTrimesh		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_dirTrimesh_setTrimesh		(int8_t *ret, void *object, void* *list, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(input,			Texture,	Texture_Input);
Dynamic_CUDA_constructTypeSkeleton(constant,		Texture,	Texture_Constant);
// Dynamic_CUDA_constructTypeSkeleton(convolutor,		Texture,	Texture_Convolutor);
// Dynamic_CUDA_constructTypeSkeleton(checkerboard,	Texture,	Texture_CheckerBoard);
// Dynamic_CUDA_constructTypeSkeleton(image,			Texture,	Texture_Image);
// Dynamic_CUDA_constructTypeSkeleton(additor,			Texture,	Texture_Math_Additor);
// Dynamic_CUDA_constructTypeSkeleton(multiplier,		Texture,	Texture_Math_Multiplier);
// Dynamic_CUDA_constructTypeSkeleton(mapSphere,		Texture,	Texture_Mapper_Sphere);
// Dynamic_CUDA_constructTypeSkeleton(dirSphere,		Texture,	Texture_Direction_Sphere);
// Dynamic_CUDA_constructTypeSkeleton(mapTrimesh,		Texture,	Texture_Mapper_Trimesh);
// Dynamic_CUDA_constructTypeSkeleton(dirTrimesh,		Texture,	Texture_Direction_Trimesh);

// Dynamic_CUDA_constructTypeConfigLinker(convolutor_setKernelSize, 	config_convolutor_setKernelSize);
// Dynamic_CUDA_constructTypeConfigLinker(convolutor_setKernelArray, 	config_convolutor_setKernelArray);
// Dynamic_CUDA_constructTypeConfigLinker(checkerboard_setBoardSize, 	config_checkerboard_setBoardSize);
// Dynamic_CUDA_constructTypeConfigLinker(additor_setAdditor, 			config_additor_setAdditor);
// Dynamic_CUDA_constructTypeConfigLinker(multiplier_setMultiplier, 	config_multiplier_setMultiplier);

// Dynamic_CUDA_constructTypeInteractLinker(convolutor_setTexture, interact_convolutor_setTexture);
// Dynamic_CUDA_constructTypeInteractLinker(mapSphere_setSphere, 	interact_mapSphere_setSphere);
// Dynamic_CUDA_constructTypeInteractLinker(dirSphere_setSphere, 	interact_dirSphere_setSphere);
// Dynamic_CUDA_constructTypeInteractLinker(mapTrimesh_setTrimesh, interact_mapTrimesh_setTrimesh);
// Dynamic_CUDA_constructTypeInteractLinker(dirTrimesh_setTrimesh, interact_dirTrimesh_setTrimesh);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Texture_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// Dynamic_CUDA_constructTypeConfigLinker(convolutor, 		convolutor_setKernelSize);
	// Dynamic_CUDA_constructTypeConfigLinker(convolutor, 		convolutor_setKernelArray);
	// Dynamic_CUDA_constructTypeConfigLinker(checkerboard, 	checkerboard_setBoardSize);
	// Dynamic_CUDA_constructTypeConfigLinker(additor, 		additor_setAdditor);
	// Dynamic_CUDA_constructTypeConfigLinker(multiplier, 		multiplier_setMultiplier);

	// Dynamic_CUDA_constructTypeInteractLinker(convolutor,	convolutor_setTexture);
	// Dynamic_CUDA_constructTypeInteractLinker(mapSphere, 	mapSphere_setSphere);
	// Dynamic_CUDA_constructTypeInteractLinker(dirSphere, 	dirSphere_setSphere);
	// Dynamic_CUDA_constructTypeInteractLinker(mapTrimesh, 	mapTrimesh_setTrimesh);
	// Dynamic_CUDA_constructTypeInteractLinker(dirTrimesh, 	dirTrimesh_setTrimesh);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(input,			input,				type_list);
	Dynamic_CUDA_addType(constant,		constant,			type_list);
	// Dynamic_CUDA_addType(convolutor,	convolutor,			type_list);
	// Dynamic_CUDA_addType(checkerboard,	checkerboard,		type_list);
	// Dynamic_CUDA_addType(image,			image,				type_list);
	// Dynamic_CUDA_addType(additor,		math_additor,		type_list);
	// Dynamic_CUDA_addType(multiplier,	math_multiplier,	type_list);
	// Dynamic_CUDA_addType(mapSphere,		mapper_sphere,		type_list);
	// Dynamic_CUDA_addType(dirSphere,		dir_sphere,			type_list);
	// Dynamic_CUDA_addType(mapTrimesh,	mapper_trimesh,		type_list);
	// Dynamic_CUDA_addType(dirTrimesh,	dir_trimesh,		type_list);
}


__host__ void RayTracer_Dynamic_Texture_info() {
}


__host__ void RayTracer_Dynamic_Texture_del() {
}


// Static Function Implementation
// table
// config
/*
__global__ static void config_convolutor_setKernelSize(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Convolutor		*texture		= (Texture_Convolutor*)object;
	int						kernel_width	= *((int*)data);

	texture->setKernel(texture->kernel, kernel_width);
	*ret = 0;
}


__global__ static void config_convolutor_setKernelArray(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Convolutor		*texture	= (Texture_Convolutor*)object;
	
	// delete original kernel
	delete[] texture->kernel;

	// create new kernel
	int32_t		kernel_size		= texture->kernel_width * texture->kernel_width;
	double		*data_double	= (double*)data;
	fp_t		*kernel			= new fp_t[kernel_size];
	for (int i = 0; i < kernel_size; i++) kernel[i] = (fp_t)(data_double[i]);

	texture->setKernel(kernel, texture->kernel_width);
	*ret = 0;
}


__global__ static void config_checkerboard_setBoardSize(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_CheckerBoard	*texture	= (Texture_CheckerBoard*)object;
	double					*vec		= (double*)data;

	texture->setBoardSize(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


__global__ static void config_additor_setAdditor(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Math_Additor	*texture	= (Texture_Math_Additor*)object;
	double					*vec		= (double*)data;

	texture->setAdditor(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


__global__ static void config_multiplier_setMultiplier(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Math_Multiplier	*texture	= (Texture_Math_Multiplier*)object;
	double					*vec		= (double*)data;

	texture->setMultiplier(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


// interact
__global__ static void interact_convolutor_setTexture(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Convolutor		*texture	= (Texture_Convolutor*)object;
	Texture					*target		= (Texture*)(list[0]);

	texture->setTexture(target);
	*ret = 0;
}


__global__ static void interact_mapSphere_setSphere(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Mapper_Sphere	*texture	= (Texture_Mapper_Sphere*)object;
	Hitable_Sphere			*sphere		= (Hitable_Sphere*)(list[0]);

	texture->setSphere(sphere);
	*ret = 0;
}


__global__ static void interact_dirSphere_setSphere(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Direction_Sphere	*texture	= (Texture_Direction_Sphere*)object;
	Hitable_Sphere				*sphere		= (Hitable_Sphere*)(list[0]);

	texture->setSphere(sphere);
	*ret = 0;
}


__global__ static void interact_mapTrimesh_setTrimesh(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Mapper_Trimesh	*texture	= (Texture_Mapper_Trimesh*)object;
	Hitable_Trimesh			*trimesh	= (Hitable_Trimesh*)(list[0]);

	texture->setTrimesh(trimesh);
	*ret = 0;

}


__global__ static void interact_dirTrimesh_setTrimesh(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Direction_Trimesh	*texture	= (Texture_Direction_Trimesh*)object;
	Hitable_Trimesh			*trimesh	= (Hitable_Trimesh*)(list[0]);

	texture->setTrimesh(trimesh);
	*ret = 0;

}
*/

#include "hip/hip_runtime.h"
#include "../inc/RayTracer_DynamicSkeleton.cuh"
#include "../inc/RayTracer_Dynamic_Texture.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// table
// __global__ static void			config_convolutor_setKernelSize		(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_convolutor_setKernelArray	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_checkerboard_setBoardSize	(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_additor_setAdditor			(int8_t *ret, void *object, uint8_t *data, uint32_t size);
// __global__ static void			config_multiplier_setMultiplier		(int8_t *ret, void *object, uint8_t *data, uint32_t size);

// __global__ static void			interact_convolutor_setTexture		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_mapSphere_setSphere		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_dirSphere_setSphere		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_mapTrimesh_setTrimesh		(int8_t *ret, void *object, void* *list, uint32_t size);
// __global__ static void			interact_dirTrimesh_setTrimesh		(int8_t *ret, void *object, void* *list, uint32_t size);

// skeleton
Dynamic_CUDA_constructTypeSkeleton(input,			Texture,	Texture_Input);
Dynamic_CUDA_constructTypeSkeleton(constant,		Texture,	Texture_Constant);
// Dynamic_CUDA_constructTypeSkeleton(convolutor,		Texture,	Texture_Convolutor);
// Dynamic_CUDA_constructTypeSkeleton(checkerboard,	Texture,	Texture_CheckerBoard);
// Dynamic_CUDA_constructTypeSkeleton(image,			Texture,	Texture_Image);
// Dynamic_CUDA_constructTypeSkeleton(additor,			Texture,	Texture_Math_Additor);
// Dynamic_CUDA_constructTypeSkeleton(multiplier,		Texture,	Texture_Math_Multiplier);
// Dynamic_CUDA_constructTypeSkeleton(mapSphere,		Texture,	Texture_Mapper_Sphere);
// Dynamic_CUDA_constructTypeSkeleton(dirSphere,		Texture,	Texture_Direction_Sphere);
// Dynamic_CUDA_constructTypeSkeleton(mapTrimesh,		Texture,	Texture_Mapper_Trimesh);
// Dynamic_CUDA_constructTypeSkeleton(dirTrimesh,		Texture,	Texture_Direction_Trimesh);

// Dynamic_CUDA_constructTypeConfigLinker(convolutor_setKernelSize, 	config_convolutor_setKernelSize);
// Dynamic_CUDA_constructTypeConfigLinker(convolutor_setKernelArray, 	config_convolutor_setKernelArray);
// Dynamic_CUDA_constructTypeConfigLinker(checkerboard_setBoardSize, 	config_checkerboard_setBoardSize);
// Dynamic_CUDA_constructTypeConfigLinker(additor_setAdditor, 			config_additor_setAdditor);
// Dynamic_CUDA_constructTypeConfigLinker(multiplier_setMultiplier, 	config_multiplier_setMultiplier);

// Dynamic_CUDA_constructTypeInteractLinker(convolutor_setTexture, interact_convolutor_setTexture);
// Dynamic_CUDA_constructTypeInteractLinker(mapSphere_setSphere, 	interact_mapSphere_setSphere);
// Dynamic_CUDA_constructTypeInteractLinker(dirSphere_setSphere, 	interact_dirSphere_setSphere);
// Dynamic_CUDA_constructTypeInteractLinker(mapTrimesh_setTrimesh, interact_mapTrimesh_setTrimesh);
// Dynamic_CUDA_constructTypeInteractLinker(dirTrimesh_setTrimesh, interact_dirTrimesh_setTrimesh);

// cuda linker function
__global__ static void	texture_addInput	(Texture *texture, Texture *input, int32_t offset);
__global__ static void	texture_rmInput		(Texture *texture, int32_t offset);
__global__ static void	texture_setPixel	(Texture *texture, fp_t point_0, fp_t point_1, fp_t point_2, fp_t pixel_0, fp_t pixel_1, fp_t pixel_2);
__global__ static void	texture_getPixel	(Texture *texture, fp_t *dst, fp_t src_0, fp_t src_1, fp_t src_2);


// Static Data
// ...


// Operation Handling
__host__ void RayTracer_Dynamic_Texture_init(std::vector<Dynamic_ContainerType*> *type_list) {
	// table
	// Dynamic_CUDA_constructTypeConfigLinker(convolutor, 		convolutor_setKernelSize);
	// Dynamic_CUDA_constructTypeConfigLinker(convolutor, 		convolutor_setKernelArray);
	// Dynamic_CUDA_constructTypeConfigLinker(checkerboard, 	checkerboard_setBoardSize);
	// Dynamic_CUDA_constructTypeConfigLinker(additor, 		additor_setAdditor);
	// Dynamic_CUDA_constructTypeConfigLinker(multiplier, 		multiplier_setMultiplier);

	// Dynamic_CUDA_constructTypeInteractLinker(convolutor,	convolutor_setTexture);
	// Dynamic_CUDA_constructTypeInteractLinker(mapSphere, 	mapSphere_setSphere);
	// Dynamic_CUDA_constructTypeInteractLinker(dirSphere, 	dirSphere_setSphere);
	// Dynamic_CUDA_constructTypeInteractLinker(mapTrimesh, 	mapTrimesh_setTrimesh);
	// Dynamic_CUDA_constructTypeInteractLinker(dirTrimesh, 	dirTrimesh_setTrimesh);

	// create type
	Dynamic_ContainerType *type;

	Dynamic_CUDA_addType(input,			input,				type_list);
	Dynamic_CUDA_addType(constant,		constant,			type_list);
	// Dynamic_CUDA_addType(convolutor,	convolutor,			type_list);
	// Dynamic_CUDA_addType(checkerboard,	checkerboard,		type_list);
	// Dynamic_CUDA_addType(image,			image,				type_list);
	// Dynamic_CUDA_addType(additor,		math_additor,		type_list);
	// Dynamic_CUDA_addType(multiplier,	math_multiplier,	type_list);
	// Dynamic_CUDA_addType(mapSphere,		mapper_sphere,		type_list);
	// Dynamic_CUDA_addType(dirSphere,		dir_sphere,			type_list);
	// Dynamic_CUDA_addType(mapTrimesh,	mapper_trimesh,		type_list);
	// Dynamic_CUDA_addType(dirTrimesh,	dir_trimesh,		type_list);
}


__host__ void RayTracer_Dynamic_Texture_info() {
}


__host__ void RayTracer_Dynamic_Texture_del() {
}


__host__ error_t Dynamic_Texture_addInput(Texture *texture, Texture *input, int32_t offset) {
	texture_addInput <<< 1, 1 >>> (texture, input, offset);
	return ERROR_NO;
}


__host__ error_t Dynamic_Texture_rmInput(Texture *texture, int32_t offset) {
	texture_rmInput <<< 1, 1 >>> (texture, offset);
	return ERROR_NO;
}


__host__ error_t Dynamic_Texture_setPixel(Texture *texture, const Vec3f &point, const Vec3f &pixel) {
	texture_setPixel <<< 1, 1 >>> (texture, point[0], point[1], point[2], pixel[0], pixel[1], pixel[2]);
	return ERROR_NO;
}


__host__ error_t Dynamic_Texture_getPixel(Texture *texture, Vec3f &dst, const Vec3f &src) {
	fp_t *dst_device;
	hipMalloc(&dst_device, 3 * sizeof(fp_t));

	texture_getPixel <<< 1, 1 >>> (texture, dst_device, src[0], src[1], src[2]);
	
	fp_t dst_host[3];
	hipMemcpy(dst_host, dst_device, 3 * sizeof(fp_t), hipMemcpyDeviceToHost);

	dst[0]	= dst_host[0];
	dst[1]	= dst_host[1];
	dst[2]	= dst_host[2];

	return ERROR_NO;
}


// Static Function Implementation
// table
// config
/*
__global__ static void config_convolutor_setKernelSize(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Convolutor		*texture		= (Texture_Convolutor*)object;
	int						kernel_width	= *((int*)data);

	texture->setKernel(texture->kernel, kernel_width);
	*ret = 0;
}


__global__ static void config_convolutor_setKernelArray(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Convolutor		*texture	= (Texture_Convolutor*)object;
	
	// delete original kernel
	delete[] texture->kernel;

	// create new kernel
	int32_t		kernel_size		= texture->kernel_width * texture->kernel_width;
	double		*data_double	= (double*)data;
	fp_t		*kernel			= new fp_t[kernel_size];
	for (int i = 0; i < kernel_size; i++) kernel[i] = (fp_t)(data_double[i]);

	texture->setKernel(kernel, texture->kernel_width);
	*ret = 0;
}


__global__ static void config_checkerboard_setBoardSize(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_CheckerBoard	*texture	= (Texture_CheckerBoard*)object;
	double					*vec		= (double*)data;

	texture->setBoardSize(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


__global__ static void config_additor_setAdditor(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Math_Additor	*texture	= (Texture_Math_Additor*)object;
	double					*vec		= (double*)data;

	texture->setAdditor(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


__global__ static void config_multiplier_setMultiplier(int8_t *ret, void *object, uint8_t *data, uint32_t size) {
	Texture_Math_Multiplier	*texture	= (Texture_Math_Multiplier*)object;
	double					*vec		= (double*)data;

	texture->setMultiplier(Vec3f(vec[0], vec[1], vec[2]));
	*ret = 0;
}


// interact
__global__ static void interact_convolutor_setTexture(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Convolutor		*texture	= (Texture_Convolutor*)object;
	Texture					*target		= (Texture*)(list[0]);

	texture->setTexture(target);
	*ret = 0;
}


__global__ static void interact_mapSphere_setSphere(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Mapper_Sphere	*texture	= (Texture_Mapper_Sphere*)object;
	Hitable_Sphere			*sphere		= (Hitable_Sphere*)(list[0]);

	texture->setSphere(sphere);
	*ret = 0;
}


__global__ static void interact_dirSphere_setSphere(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Direction_Sphere	*texture	= (Texture_Direction_Sphere*)object;
	Hitable_Sphere				*sphere		= (Hitable_Sphere*)(list[0]);

	texture->setSphere(sphere);
	*ret = 0;
}


__global__ static void interact_mapTrimesh_setTrimesh(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Mapper_Trimesh	*texture	= (Texture_Mapper_Trimesh*)object;
	Hitable_Trimesh			*trimesh	= (Hitable_Trimesh*)(list[0]);

	texture->setTrimesh(trimesh);
	*ret = 0;

}


__global__ static void interact_dirTrimesh_setTrimesh(int8_t *ret, void *object, void* *list, uint32_t size) {
	Texture_Direction_Trimesh	*texture	= (Texture_Direction_Trimesh*)object;
	Hitable_Trimesh			*trimesh	= (Hitable_Trimesh*)(list[0]);

	texture->setTrimesh(trimesh);
	*ret = 0;

}
*/


// cuda linker function
__global__ static void texture_addInput(Texture *texture, Texture *input, int32_t offset) {
	texture->addInput(input, offset);
}


__global__ static void texture_rmInput(Texture *texture, int32_t offset) {
	texture->rmInput(offset);
}


__global__ static void texture_setPixel(Texture *texture, fp_t point_0, fp_t point_1, fp_t point_2, fp_t pixel_0, fp_t pixel_1, fp_t pixel_2) {
	texture->setPixel(
		Vec3f(point_0, point_1, point_2),
		Vec3f(pixel_0, pixel_1, pixel_2));
}


__global__ static void texture_getPixel(Texture *texture, fp_t *dst, fp_t src_0, fp_t src_1, fp_t src_2) {
	Vec3f vec_pixel;
	texture->getPixel(vec_pixel, Vec3f(src_0, src_1, src_2));

	dst[0]	= vec_pixel[0];
	dst[1]	= vec_pixel[1];
	dst[2]	= vec_pixel[2];
}

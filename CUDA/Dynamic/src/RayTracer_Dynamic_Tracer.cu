#include "hip/hip_runtime.h"
#include <stdint.h>
#include "../../Tracer/Tracer.cuh"
#include "../inc/RayTracer_DynamicContainer.cuh"
#include "../inc/RayTracer_Dynamic_Tracer.cuh"

// TEST
#include "../inc/RayTracer_Dynamic_Scene.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
__host__ static void			fix_index			(int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, int w, int h, int is_reverse_x, int is_reverse_y);

__host__ static inline void		host_init			();
__host__ static inline Vec3f	host_trace			(Camera *camera, fp_t x, fp_t y, int depth);

__global__ static void			global_init			();
__global__ static void			global_trace		(fp_t *value, void *camera, fp_t x, fp_t y, int depth);


// Static Data
__device__	RayTracer					*tracer;
__device__	Scene						*scene;
extern Dynamic_ContainerList<Camera>	camera_list;


// Operation Handling
__host__ void RayTracer_Dynamic_Tracer_init() {
	host_init();
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB888(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	uint8_t *pixel = (uint8_t*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return ERROR_ANY;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);

			// TODO: should it be uint8 instead of int ?
			pixel[index + 0] = (int)(result[0] * 255);
			pixel[index + 1] = (int)(result[1] * 255);
			pixel[index + 2] = (int)(result[2] * 255);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return ERROR_NO;
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB64F(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	double *pixel = (double*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (double)(result[0]);
			pixel[index + 1] = (double)(result[1]);
			pixel[index + 2] = (double)(result[2]);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
}


// Static Function Implementation
__host__ static void fix_index(
	int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
	int w, int h, int is_reverse_x, int is_reverse_y) {

	// reverse x or not
	if (!is_reverse_x) {
		*index_x_start	= 0;
		*index_x_next	= 1;
	} else {
		*index_x_start	= w - 1;
		*index_x_next	= -1;
	}

	// reverse y or not
	if (!is_reverse_y) {
		*index_y_start	= 0;
		*index_y_next	= 1;
	} else {
		*index_y_start	= h - 1;
		*index_y_next	= -1;
	}
}


// host
__host__ static inline void host_init() {
	global_init <<< 1, 1 >>> ();
}


__host__ static Vec3f host_trace(Camera *camera, fp_t x, fp_t y, int depth) {
	// create device memory space for resultant pixel
	fp_t *pixel_device;
	hipMalloc(&pixel_device, 3 * sizeof(fp_t));

	// kernel operation
	global_trace<<< 1, 1 >>>(pixel_device, camera, x, y, depth);

	// get resultant pixel
	fp_t pixel_host[3];
	hipMemcpy(pixel_host, pixel_device, 3 * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipFree(pixel_device);

	// printf("%f %f %f \n", pixel_host[0], pixel_host[1], pixel_host[2]);

	return Vec3f(pixel_host[0], pixel_host[1], pixel_host[2]);
}


// global
__global__ static void global_init() {
	tracer	= new RayTracer();
	scene	= new Scene();

	// tracer
	tracer->setScene(scene);

	// scene
	// allocate space
	int32_t size_hitable	= 1000;
	int32_t size_light		= 100;

	void	*hitable_list;
	void	*light_list;

	hipMalloc(&hitable_list,	size_hitable * sizeof(SceneObject_Hitable));
	hipMalloc(&light_list,		size_light * sizeof(SceneObject_Light));

	scene->allocateHitable(hitable_list, size_hitable);
	scene->allocateLight(light_list, size_light);
}


__global__ static void global_trace(fp_t *value, void *camera, fp_t x, fp_t y, int depth) {
	Vec3f result = tracer->trace((Camera*)camera, x, y, depth);

	// TODO: remove
	// Camera *c = (Camera*)camera;
	// printf(
	// 	"%f %f %f %f %f %f %f\n",
	// 	c->look_from[0], c->look_from[1], c->look_from[2],
	// 	c->look_at[0], c->look_at[1], c->look_at[2],
	// 	c->aspect);

	value[0] = result[0];
	value[1] = result[1];
	value[2] = result[2];
}

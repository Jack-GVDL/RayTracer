#include "hip/hip_runtime.h"
#include <stdint.h>
#include "../../Tracer/Tracer.cuh"
#include "../inc/RayTracer_DynamicContainer.cuh"
#include "../inc/RayTracer_Dynamic_Tracer.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
__host__ static void	fix_index		(int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, int w, int h, int is_reverse_x, int is_reverse_y);

__host__ static inline void		host_init			();
__host__ static inline Vec3f	host_trace			(Camera *camera, fp_t x, fp_t y, int depth);
__host__ static inline int8_t	host_add_light		(SceneObject_Light *light);
__host__ static inline int8_t	host_add_hitable	(SceneObject_Hitable *hitable);
__host__ static inline int8_t	host_rm_light		(SceneObject_Light *light);
__host__ static inline int8_t	host_rm_hitable		(SceneObject_Hitable *hitable);

__global__ static void			global_init			();
__global__ static void			global_add_light	(SceneObject_Light *light);
__global__ static void			global_add_hitable	(SceneObject_Hitable *hitable);
__global__ static void			global_rm_light		(SceneObject_Light *light);
__global__ static void			global_rm_hitable	(SceneObject_Hitable *hitable);
__global__ static void			global_trace		(fp_t *value, void *camera, fp_t x, fp_t y, int depth);


// Static Data
__device__	RayTracer					*tracer;
__device__	Scene						*scene;
extern	Dynamic_ContainerList<Camera>	camera_list;


// Operation Handling
__host__ void RayTracer_Dynamic_Tracer_init() {
	host_init();
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB888(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	uint8_t *pixel = (uint8_t*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (int)(result[0] * 255);
			pixel[index + 1] = (int)(result[1] * 255);
			pixel[index + 2] = (int)(result[2] * 255);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB64F(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	double *pixel = (double*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (double)(result[0]);
			pixel[index + 1] = (double)(result[1]);
			pixel[index + 2] = (double)(result[2]);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
}


__host__ int8_t RayTracer_Dynamic_Scene_addHitable(SceneObject_Hitable *hitable) {
	return host_add_hitable(hitable);
}


__host__ int8_t RayTracer_Dynamic_Scene_addLight(SceneObject_Light *light) {
	return host_add_light(light);
}


__host__ int8_t RayTracer_Dynamic_Scene_rmHitable(SceneObject_Hitable *hitable) {
	return host_rm_hitable(hitable);
}


__host__ int8_t RayTracer_Dynamic_Scene_rmLight(SceneObject_Light *light) {
	return host_rm_light(light);
}


// Static Function Implementation
__host__ static void fix_index(
	int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
	int w, int h, int is_reverse_x, int is_reverse_y) {

	// reverse x or not
	if (!is_reverse_x) {
		*index_x_start	= 0;
		*index_x_next	= 1;
	} else {
		*index_x_start	= w - 1;
		*index_x_next	= -1;
	}

	// reverse y or not
	if (!is_reverse_y) {
		*index_y_start	= 0;
		*index_y_next	= 1;
	} else {
		*index_y_start	= h - 1;
		*index_y_next	= -1;
	}
}


// host
__host__ static inline void host_init() {
	global_init <<< 1, 1 >>> ();
}


__host__ static inline int8_t host_add_light(SceneObject_Light *light) {
	global_add_light <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ static inline int8_t host_add_hitable(SceneObject_Hitable *hitable) {
	global_add_hitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


__host__ static inline int8_t host_rm_light(SceneObject_Light *light) {
	global_rm_light <<< 1, 1 >>> (light);
	return ERROR_NO;
}


__host__ static inline int8_t host_rm_hitable(SceneObject_Hitable *hitable) {
	global_rm_hitable <<< 1, 1 >>> (hitable);
	return ERROR_NO;
}


__host__ static Vec3f host_trace(Camera *camera, fp_t x, fp_t y, int depth) {
	// create device memory space for resultant pixel
	fp_t *pixel_device;
	hipMalloc(&pixel_device, 3 * sizeof(fp_t));

	// kernel operation
	global_trace<<< 1, 1 >>>(pixel_device, camera, x, y, depth);

	// get resultant pixel
	fp_t pixel_host[3];
	hipMemcpy(pixel_host, pixel_device, 3 * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipFree(pixel_device);

	return Vec3f(pixel_host[0], pixel_host[1], pixel_host[2]);
}


// global
__global__ static void global_init() {
	tracer	= new RayTracer();
	scene	= new Scene();

	// tracer
	tracer->setScene(scene);

	// scene
	// allocate space
	int32_t size_hitable	= 1000;
	int32_t size_light		= 100;

	void	*hitable_list;
	void	*light_list;

	hipMalloc(&hitable_list,	size_hitable * sizeof(SceneObject_Hitable));
	hipMalloc(&light_list,		size_light * sizeof(SceneObject_Light));

	scene->allocateHitable(hitable_list, size_hitable);
	scene->allocateLight(light_list, size_light);
}


__global__ static void global_add_light(SceneObject_Light *light) {
	scene->addLight(light);
}


__global__ static void global_add_hitable(SceneObject_Hitable *hitable) {
	scene->addHitable(hitable);
}


__global__ static void global_rm_light(SceneObject_Light *light) {
	scene->rmLight(light);
}


__global__ static void global_rm_hitable(SceneObject_Hitable *hitable) {
	scene->rmHitable(hitable);
}


__global__ static void global_trace(fp_t *value, void *camera, fp_t x, fp_t y, int depth) {
	Vec3f result = tracer->trace((Camera*)camera, x, y, depth);

	value[0] = result[0];
	value[1] = result[1];
	value[2] = result[2];
}

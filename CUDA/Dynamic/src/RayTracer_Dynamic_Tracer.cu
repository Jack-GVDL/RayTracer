#include "hip/hip_runtime.h"
#include <stdint.h>
#include "../../Tracer/Tracer.cuh"
#include "../inc/RayTracer_DynamicContainer.cuh"
#include "../inc/RayTracer_Dynamic_Tracer.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
__host__ static void	fix_index		(int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, int w, int h, int is_reverse_x, int is_reverse_y);
__host__ static Vec3f	host_trace		(Camera *camera, fp_t x, fp_t y, int depth);
__global__ static void	global_trace	(fp_t *value, void *camera, fp_t *x, fp_t *y, int depth);


// Static Data
extern	RayTracer						tracer;
extern	Dynamic_ContainerList<Camera>	camera_list;


// Operation Handling
__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB888(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	uint8_t *pixel = (uint8_t*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (int)(result[0] * 255);
			pixel[index + 1] = (int)(result[1] * 255);
			pixel[index + 2] = (int)(result[2] * 255);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB64F(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	// pointer casting
	double *pixel = (double*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (double)(result[0]);
			pixel[index + 1] = (double)(result[1]);
			pixel[index + 2] = (double)(result[2]);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
}


// Static Function Implementation
__host__ static void fix_index(
	int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
	int w, int h, int is_reverse_x, int is_reverse_y) {

	// reverse x or not
	if (!is_reverse_x) {
		*index_x_start	= 0;
		*index_x_next	= 1;
	} else {
		*index_x_start	= w - 1;
		*index_x_next	= -1;
	}

	// reverse y or not
	if (!is_reverse_y) {
		*index_y_start	= 0;
		*index_y_next	= 1;
	} else {
		*index_y_start	= h - 1;
		*index_y_next	= -1;
	}
}


__host__ static Vec3f host_trace(Camera *camera, fp_t x, fp_t y, int depth) {
	
}


__global__ static void global_trace(fp_t *value, void *camera, fp_t *x, fp_t *y, int depth) {

}

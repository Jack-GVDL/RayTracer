#include "hip/hip_runtime.h"
#include <stdint.h>
#include "../../Tracer/Tracer.cuh"
#include "../inc/RayTracer_DynamicContainer.cuh"
#include "../inc/RayTracer_Dynamic_Tracer.cuh"


// Define
#define SIZE_SCHEDULER	128
#define SIZE_BLOCK		2
#define SIZE_THREAD		64


// Typedef
// ...


// Static Function Prototype
__host__ static void			fix_index						(int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
																 int w, int h, int is_reverse_x, int is_reverse_y);

__host__ static inline void		host_init						();
__host__ static inline void		host_Tracer_trace				(Camera *camera, fp_t *buffer_list, fp_t *position_list, int32_t size, int32_t depth);
__host__ static inline void		host_Tracer_getPosition			(fp_t *position_list, int32_t w, int32_t h, 
																 int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next);

__global__ static void			global_init						();
__global__ static void			global_Tracer_addScheduler		(int32_t memory_size);
__global__ static void			global_Tracer_trace				(void *camera, fp_t *dst, fp_t *position_list, int32_t offset, int32_t depth);
__global__ static void			global_Tracer_getPosition		(fp_t *position_list, int32_t w, int32_t h,
																 int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next,
																 int32_t offset);
// backup
// __global__ static void			global_Tracer_resetScheduler	();


// Static Data
__device__	Tracer						*tracer;
__device__	Scene						*scene;
extern Dynamic_ContainerList<Camera>	camera_list;


// Operation Handling
__host__ void RayTracer_Dynamic_Tracer_init() {
	host_init();
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB888(
	int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {

	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) return ERROR_ANY;

	// allocate for buffer and position
	fp_t *position_device;
	fp_t *buffer_host;
	fp_t *buffer_device;

	buffer_host = (fp_t*)malloc(w * h * 3 * sizeof(fp_t));

	hipMalloc(&position_device,	w * h * 2 * sizeof(fp_t));
	hipMalloc(&buffer_device,		w * h * 3 * sizeof(fp_t));

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// get position list
	host_Tracer_getPosition(position_device, w, h, index_x_start, index_y_start, index_x_next, index_y_next);

	// get trace result
	host_Tracer_trace(container_camera->getObject(), buffer_device, position_device, w * h, depth);
	hipMemcpy(buffer_host, buffer_device, w * h * 3 * sizeof(fp_t), hipMemcpyDeviceToHost);

	// convert it from fp_t [0, 1] to RGB888 [0, 255]
	uint8_t *pixel = (uint8_t*)data;
	for (int32_t i = 0; i < w * h * 3; ++i) {
		pixel[i] = (uint8_t)(buffer_host[i] * 255);
	}

	// free space
	free(buffer_host);
	hipFree(position_device);
	hipFree(buffer_device);

	return ERROR_NO;
}


// TODO: not yet completed, wait for above code to be finalized
__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB64F(
	int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) return ERROR_ANY;

	// allocate for buffer and position
	fp_t *position_device;
	fp_t *buffer_host;
	fp_t *buffer_device;

	buffer_host = (fp_t*)malloc(w * h * 3 * sizeof(fp_t));

	hipMalloc(&position_device,	w * h * 2 * sizeof(fp_t));
	hipMalloc(&buffer_device,		w * h * 3 * sizeof(fp_t));

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// get position list
	host_Tracer_getPosition(position_device, w, h, index_x_start, index_y_start, index_x_next, index_y_next);

	// get trace result
	host_Tracer_trace(container_camera->getObject(), buffer_device, position_device, w * h, depth);
	hipMemcpy(buffer_host, buffer_device, w * h * 3 * sizeof(fp_t), hipMemcpyDeviceToHost);

	// convert it from fp_t to double
	double *pixel = (double*)data;
	for (int32_t i = 0; i < w * h * 3; ++i) {
		pixel[i] = (double)(buffer_host[i]);
	}

	// free space
	free(buffer_host);
	hipFree(position_device);
	hipFree(buffer_device);

	return ERROR_NO;
}



// TODO: should it be exposed or not ?
__host__ error_t Dynamic_Tracer_addScheduler(int32_t size_memory) {
	global_Tracer_addScheduler <<< 1, 1 >>> (size_memory);
	return ERROR_NO;
}


// backup
/*
__host__ error_t Dynamic_Tracer_resetScheduler() {
	global_Tracer_resetScheduler <<< 1, 1 >>> ();
	return ERROR_NO;
}
*/


// Static Function Implementation
__host__ static void fix_index(
	int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
	int w, int h, int is_reverse_x, int is_reverse_y) {

	// reverse x or not
	if (!is_reverse_x) {
		*index_x_start	= 0;
		*index_x_next	= 1;
	} else {
		*index_x_start	= w - 1;
		*index_x_next	= -1;
	}

	// reverse y or not
	if (!is_reverse_y) {
		*index_y_start	= 0;
		*index_y_next	= 1;
	} else {
		*index_y_start	= h - 1;
		*index_y_next	= -1;
	}
}


// host
__host__ static inline void host_init() {
	// device side
	global_init <<< 1, 1 >>> ();

	// TODO: currently number of scheduler is fixed
	for (int32_t i = 0; i < SIZE_SCHEDULER; ++i) Dynamic_Tracer_addScheduler(1024 * 5 * sizeof(uint8_t));
}


__host__ static inline void host_Tracer_getPosition(
	fp_t *position_list, int32_t w, int32_t h, 
	int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next) {

	global_Tracer_getPosition <<< 1, 100 >>> (
		position_list, w, h, index_x_start, index_y_start, index_x_next, index_y_next, 100);
}


__host__ static inline void host_Tracer_trace(
	Camera *camera, fp_t *buffer_list, fp_t *position_list, int32_t size, int32_t depth) {

	// TODO: currently number of scheduler is fixed
	const int32_t	offset	= SIZE_SCHEDULER;
	int32_t			index	= 0;

	for (int32_t i = 0; i < (size / offset); ++i) {
		global_Tracer_trace <<< 1, SIZE_SCHEDULER >>> (camera, buffer_list, position_list, index, depth);
		index += offset;
	}
}


// global
__global__ static void global_init() {
	tracer	= new Tracer();
	scene	= new Scene();

	// tracer
	tracer->setScene(scene);

	// scene
	// allocate space
	int32_t size_hitable	= 1000;
	int32_t size_light		= 100;

	void	*hitable_list;
	void	*light_list;

	hipMalloc(&hitable_list,	size_hitable * sizeof(SceneObject_Hitable));
	hipMalloc(&light_list,		size_light * sizeof(SceneObject_Light));

	scene->allocateHitable(hitable_list, size_hitable);
	scene->allocateLight(light_list, size_light);
}


__global__ static void global_Tracer_addScheduler(int32_t memory_size) {
	// allocate space
	void *memory = nullptr;
	hipMalloc(&memory, memory_size);

	Scheduler_Scatter *scheduler = new Scheduler_Scatter();
	// hipMalloc(&scheduler, sizeof(Scheduler_Scatter*));

	// config scheduler
	scheduler->memory_control.setMemory(memory, memory_size);

	// add to tracer
	tracer->addScheduler(scheduler);
}


// backup
/*
__global__ static void global_Tracer_resetScheduler() {
	tracer->resetScheduler();
}
*/


__global__ static void global_Tracer_trace(void *camera, fp_t *dst, fp_t *position_list, int32_t offset, int32_t depth) {
	// get thread-block index (global index)
	// just use int type
	int global_index = blockIdx.x * blockDim.x + threadIdx.x;

	// get pixel intensity
	Vec3f result = tracer->trace(
		(Camera*)camera,
		position_list[(offset + global_index) * 2 + 0],
		position_list[(offset + global_index) * 2 + 1],
		depth, global_index);
	
	// set to dst
	dst[(offset + global_index) * 3 + 0] = result[0];
	dst[(offset + global_index) * 3 + 1] = result[1];
	dst[(offset + global_index) * 3 + 2] = result[2];
}


__global__ static void global_Tracer_getPosition(
	fp_t *position_list, int32_t w, int32_t h, 
	int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next,
	int32_t offset) {

	int global_index = blockIdx.x * blockDim.x + threadIdx.x;

	// first calculate x_half and y_half to save time for later work
	fp_t x_half = fp_t(w) / 2.0;
	fp_t y_half = fp_t(h) / 2.0;

	// loop through all pixel
	// find the corresponding x and y on camera for each output image pixel
	int32_t	index_y	= index_y_start;

	for (int y = 0; y < h; y++) {
		
		for (int x = 0; x < w; x += offset) {
			if (x + global_index >= w) continue;

			const fp_t u = ((fp_t)(index_x_start + index_x_next * (x + global_index)) - x_half) / x_half;
			const fp_t v = ((fp_t)(index_y) - y_half) / y_half;
			
			position_list[(y * w + x + global_index) * 2 + 0] = u;
			position_list[(y * w + x + global_index) * 2 + 1] = v;
		}

		index_y += index_y_next;
	}
}

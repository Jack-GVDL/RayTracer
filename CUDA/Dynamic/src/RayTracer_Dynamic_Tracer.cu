#include "hip/hip_runtime.h"
#include <stdint.h>
#include "../../Tracer/Tracer.cuh"
#include "../inc/RayTracer_DynamicContainer.cuh"
#include "../inc/RayTracer_Dynamic_Tracer.cuh"

// TEST
#include "../inc/RayTracer_Dynamic_Scene.cuh"


// Define
#define SIZE_SCHEDULER	100


// Typedef
// ...


// Static Function Prototype
__host__ static void			get_position					(fp_t *position_list, int32_t w, int32_t h, 
																 int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next);

__host__ static void			fix_index						(int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
																 int w, int h, int is_reverse_x, int is_reverse_y);

__host__ static inline void		host_init						();
__host__ static inline void		host_Tracer_trace				(Camera *camera, fp_t *buffer_list, fp_t *position_list, int32_t size, int32_t depth);

__global__ static void			global_init						();
__global__ static void			global_Tracer_addScheduler		(int32_t memory_size);
// backup
// __global__ static void			global_Tracer_resetScheduler	();
__global__ static void			global_Tracer_trace				(void *camera, fp_t *dst, fp_t *position_list, int32_t offset, int32_t depth);

// TODO: remove
// __global__ static void			global_Tracer_trace				(fp_t *value, void *camera, fp_t x, fp_t y, int32_t depth, int32_t index);


// Static Data
__device__	Tracer						*tracer;
__device__	Scene						*scene;
extern Dynamic_ContainerList<Camera>	camera_list;


// Operation Handling
__host__ void RayTracer_Dynamic_Tracer_init() {
	host_init();
}


__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB888(
	int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {

	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) return ERROR_ANY;

	// allocate for buffer and position
	fp_t *position_host;
	fp_t *position_device;

	fp_t *buffer_host;
	fp_t *buffer_device;

	position_host	= (fp_t*)malloc(w * h * 2 * sizeof(fp_t));
	buffer_host		= (fp_t*)malloc(w * h * 3 * sizeof(fp_t));

	hipMalloc(&position_device,	w * h * 2 * sizeof(fp_t));
	hipMalloc(&buffer_device,		w * h * 3 * sizeof(fp_t));

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// get position list
	get_position(position_host, w, h, index_x_start, index_y_start, index_x_next, index_y_next);
	hipMemcpy(position_device, position_host, w * h * 2 * sizeof(fp_t), hipMemcpyHostToDevice);

	// get trace result
	host_Tracer_trace(container_camera->getObject(), buffer_device, position_device, w * h, depth);
	hipMemcpy(buffer_host, buffer_device, w * h * 3 * sizeof(fp_t), hipMemcpyDeviceToHost);

	// convert it from fp_t [0, 1] to RGB888 [0, 255]
	uint8_t *pixel = (uint8_t*)data;
	for (int32_t i = 0; i < w * h * 3; ++i) {
		pixel[i] = (uint8_t)(buffer_host[i] * 255);
	}

	// free space
	free(position_host);
	free(buffer_host);
	hipFree(position_device);
	hipFree(buffer_device);

	return ERROR_NO;
}


// TODO: not yet completed, wait for above code to be finalized
__host__ int RayTracer_Dynamic_Tracer_tracePoint_RGB64F(int index_camera, void *data, int w, int h, int depth, int is_reverse_x, int is_reverse_y) {
	/*
	// pointer casting
	double *pixel = (double*)data;
	
	// get camera
	Dynamic_Container<Camera> *container_camera = camera_list.get(index_camera);
	if (container_camera == nullptr) {
		pixel[0] = 0;
		pixel[1] = 0;
		pixel[2] = 0;
		return -1;
	}

	// based on property of export image
	// order of putting pixel on the linear memory space may be different
	int index_x_start, 		index_y_start;
	int index_x_next, 		index_y_next;
	fix_index(&index_x_start, &index_y_start, &index_x_next, &index_y_next, w, h, is_reverse_x, is_reverse_y);

	// first calculate x_half and y_half to save time for later work
	double x_half = double(w) / 2.0;
	double y_half = double(h) / 2.0;

	// loop through all pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const double u = (double(index_x) - x_half) / x_half;
			const double v = (double(index_y) - y_half) / y_half;

			Vec3f result = host_trace(container_camera->getObject(), u, v, depth);
			pixel[index + 0] = (double)(result[0]);
			pixel[index + 1] = (double)(result[1]);
			pixel[index + 2] = (double)(result[2]);

			index_x += index_x_next;
			index	+= 3;
		}

		index_y	+= index_y_next;
	}

	return 0;
	*/

	return ERROR_NO;
}


__host__ error_t Dynamic_Tracer_addScheduler(int32_t size_memory) {
	global_Tracer_addScheduler <<< 1, 1 >>> (size_memory);
	return ERROR_NO;
}


// backup
/*
__host__ error_t Dynamic_Tracer_resetScheduler() {
	global_Tracer_resetScheduler <<< 1, 1 >>> ();
	return ERROR_NO;
}
*/


// Static Function Implementation
__host__ static void get_position(
	fp_t *position_list, int32_t w, int32_t h, 
	int32_t index_x_start, int32_t index_y_start, int32_t index_x_next, int32_t index_y_next) {

	// first calculate x_half and y_half to save time for later work
	fp_t x_half = fp_t(w) / 2.0;
	fp_t y_half = fp_t(h) / 2.0;

	// loop through all pixel
	// find the corresponding x and y on camera for each output image pixel
	int index	= 0;
	int index_x	= index_x_start;
	int index_y	= index_y_start;

	for (int y = 0; y < h; y++) {

		index_x = index_x_start;
		for (int x = 0; x < w; x++) {

			const fp_t u = (double(index_x) - x_half) / x_half;
			const fp_t v = (double(index_y) - y_half) / y_half;

			position_list[index + 0] = u;
			position_list[index + 1] = v;

			index_x += index_x_next;
			index	+= 2;
		}

		index_y	+= index_y_next;
	}
}


__host__ static void fix_index(
	int *index_x_start, int *index_y_start, int *index_x_next, int *index_y_next, 
	int w, int h, int is_reverse_x, int is_reverse_y) {

	// reverse x or not
	if (!is_reverse_x) {
		*index_x_start	= 0;
		*index_x_next	= 1;
	} else {
		*index_x_start	= w - 1;
		*index_x_next	= -1;
	}

	// reverse y or not
	if (!is_reverse_y) {
		*index_y_start	= 0;
		*index_y_next	= 1;
	} else {
		*index_y_start	= h - 1;
		*index_y_next	= -1;
	}
}


// host
__host__ static inline void host_init() {
	// device side
	global_init <<< 1, 1 >>> ();

	// TODO: currently number of scheduler is fixed
	for (int32_t i = 0; i < SIZE_SCHEDULER; ++i) Dynamic_Tracer_addScheduler(1024 * 5 * sizeof(uint8_t));
}


// TODO: remove
/*
__host__ static Vec3f host_trace(Camera *camera, fp_t x, fp_t y, int depth) {
	// create device memory space for resultant pixel
	fp_t *pixel_device;
	hipMalloc(&pixel_device, 3 * sizeof(fp_t));

	// kernel operation
	global_Tracer_trace<<< 1, 1 >>>(pixel_device, camera, x, y, depth);

	// get resultant pixel
	fp_t pixel_host[3];
	hipMemcpy(pixel_host, pixel_device, 3 * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipFree(pixel_device);

	return Vec3f(pixel_host[0], pixel_host[1], pixel_host[2]);
}
*/


__host__ static inline void host_Tracer_trace(
	Camera *camera, fp_t *buffer_list, fp_t *position_list, int32_t size, int32_t depth) {

	// TODO: currently number of scheduler is fixed
	const int32_t	offset	= SIZE_SCHEDULER;
	int32_t			index	= 0;

	for (int32_t i = 0; i < (size / offset); ++i) {
		global_Tracer_trace <<< 1, SIZE_SCHEDULER >>> (camera, buffer_list, position_list, index, depth);
		index += offset;
	}
}


// global
__global__ static void global_init() {
	tracer	= new Tracer();
	scene	= new Scene();

	// tracer
	tracer->setScene(scene);

	// scene
	// allocate space
	int32_t size_hitable	= 1000;
	int32_t size_light		= 100;

	void	*hitable_list;
	void	*light_list;

	hipMalloc(&hitable_list,	size_hitable * sizeof(SceneObject_Hitable));
	hipMalloc(&light_list,		size_light * sizeof(SceneObject_Light));

	scene->allocateHitable(hitable_list, size_hitable);
	scene->allocateLight(light_list, size_light);
}


__global__ static void global_Tracer_addScheduler(int32_t memory_size) {
	// allocate space
	void *memory = nullptr;
	hipMalloc(&memory, memory_size);

	Scheduler_Scatter *scheduler = new Scheduler_Scatter();
	// hipMalloc(&scheduler, sizeof(Scheduler_Scatter*));

	// config scheduler
	scheduler->memory_control.setMemory(memory, memory_size);

	// add to tracer
	tracer->addScheduler(scheduler);
}


// backup
/*
__global__ static void global_Tracer_resetScheduler() {
	tracer->resetScheduler();
}
*/


// TODO: remove
/*
__global__ static void global_Tracer_trace(fp_t *value, void *camera, fp_t x, fp_t y, int32_t depth, int32_t index) {
	Vec3f result = tracer->trace((Camera*)camera, x, y, depth);

	value[0] = result[0];
	value[1] = result[1];
	value[2] = result[2];
}
*/


// TODO: not yet completed
__global__ static void global_Tracer_trace(void *camera, fp_t *dst, fp_t *position_list, int32_t offset, int32_t depth) {
	// get thread-block index (global index)
	// just use int type
	int global_index = blockIdx.x * blockDim.x + threadIdx.x;

	// get pixel intensity
	Vec3f result = tracer->trace(
		(Camera*)camera,
		position_list[(offset + global_index) * 2 + 0],
		position_list[(offset + global_index) * 2 + 1],
		depth, global_index);

	// TEST
	// if (result[0] != 0 || result[1] != 0 || result[2] != 0) printf("%f %f %f \n", result[0], result[1], result[2]);

	// set to dst
	dst[(offset + global_index) * 3 + 0] = result[0];
	dst[(offset + global_index) * 3 + 1] = result[1];
	dst[(offset + global_index) * 3 + 2] = result[2];
}

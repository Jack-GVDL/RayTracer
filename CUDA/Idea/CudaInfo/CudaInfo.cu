#include <stdio.h>
#include <hip/hip_runtime.h>



__host__ int main(void) {
	int				dev		= 0;
	hipDeviceProp_t 	dev_prop;

	hipGetDeviceProperties(&dev_prop, dev);

	printf("GPU Device              %s \n",		dev_prop.name);
	printf("SM                      %i \n",		dev_prop.multiProcessorCount);
	printf("Shared Memory per Block %i KB \n",	dev_prop.sharedMemPerBlock / 1024);
	printf("Max Thread per Block    %i \n",		dev_prop.maxThreadsPerBlock);
	printf("Max Thread per SM       %i \n",		dev_prop.maxThreadsPerMultiProcessor);
	printf("Max WARP per SM         %i \n",		dev_prop.maxThreadsPerMultiProcessor / 32);

	return 0;
}

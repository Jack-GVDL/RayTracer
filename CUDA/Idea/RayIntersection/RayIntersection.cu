#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "RayTracer_Vec3f.hpp"


typedef float fp_t;
// typedef float3 Vec3f_Device;


template <class T>
inline __device__ T max(T x, T y) {
	return x > y ? x : y;
}


template <class T>
inline __device__ T min(T x, T y) {
	return  x < y ? x : y;
}


// Data Structure
class Vec3f_Device {
	// Data
	public:
		fp_t n[3];

	// Operation
	public:
		// init
		__device__ Vec3f_Device() { 
			n[0] = 0.0; 
			n[1] = 0.0; 
			n[2] = 0.0;
		}

		__device__ Vec3f_Device(fp_t n0) {
			n[0] = n0;
			n[1] = n0;
			n[2] = n0;
		}

		__device__ Vec3f_Device(fp_t n0, fp_t n1, fp_t n2) {
			n[0] = n0; 
			n[1] = n1; 
			n[2] = n2;
		}

		__device__ Vec3f_Device(const Vec3f_Device &v) {
			n[0] = v.n[0];
			n[1] = v.n[1];
			n[2] = v.n[2];
		}

		// operator
		__device__	fp_t			operator	[]	( int i )			const { return n[i]; }
		__device__	fp_t&			operator	[]	( int i )			{ return n[i]; }

		__device__	const Vec3f_Device&	operator	+	()					const { return *this; }
		__device__	Vec3f_Device			operator	-	()					const { return Vec3f_Device(-n[0], -n[1], -n[2]); }

		__device__	Vec3f_Device&			operator	=	( const Vec3f_Device &v )	{ n[0] = v.n[0]; n[1] = v.n[1]; n[2] = v.n[2]; return *this; }
		__device__	Vec3f_Device&			operator	+=	( const Vec3f_Device &v )	{ n[0] += v.n[0]; n[1] += v.n[1]; n[2] += v.n[2]; return *this; }
		__device__	Vec3f_Device&			operator	-=	( const Vec3f_Device &v )	{ n[0] -= v.n[0]; n[1] -= v.n[1]; n[2] -= v.n[2]; return *this; }
		__device__	Vec3f_Device&			operator	*=	( const Vec3f_Device &v )	{ n[0] *= v.n[0]; n[1] *= v.n[1]; n[2] *= v.n[2]; return *this; }
		__device__	Vec3f_Device&			operator	/=	( const Vec3f_Device &v )	{ n[0] /= v.n[0]; n[1] /= v.n[1]; n[2] /= v.n[2]; return *this; }

		__device__	Vec3f_Device&			operator	*=	( const fp_t d )	{ n[0] *= d; n[1] *= d; n[2] *= d; return *this; }
		__device__	Vec3f_Device&			operator	/=	( const fp_t d )	{ n[0] /= d; n[1] /= d; n[2] /= d; return *this; }

		// Math Operation
		// TODO: clamp (0.0 <= n[i] <= 1.0)

		Vec3f_Device __device__ clamp(fp_t val_min, fp_t val_max) const {
			return Vec3f_Device(
				max(val_min, min(n[0], val_max)),
				max(val_min, min(n[1], val_max)),
				max(val_min, min(n[2], val_max)));
		}

		bool __device__ isZero() const {
			return n[0] == 0 && n[1] == 0 && n[2] == 0;
		}

		Vec3f_Device __device__ cross(const Vec3f_Device &v) const {
			return Vec3f_Device(
			n[1] * v[2] - n[2] * v[1],
			n[2] * v[0] - n[0] * v[2],
			n[0] * v[1] - n[1] * v[0] );
		}

		fp_t __device__ dot(const Vec3f_Device &v) const {
			return n[0] * v[0] + n[1] * v[1] + n[2] * v[2];
		}

		Vec3f_Device __device__ prod(const Vec3f_Device &v) const {
			return Vec3f_Device(n[0] * v[0], n[1] * v[1], n[2] * v[2]);
		}

		// reference
		// 1. https://en.wikipedia.org/wiki/Vector_projection
		fp_t __device__ projectLength(const Vec3f_Device &v) const {
			fp_t dot_result	= dot(v);
			fp_t length_v2	= v.lengthSquared();
			return dot_result / length_v2;
		}

		// reference
		// 1. https://en.wikipedia.org/wiki/Vector_projection
		// projection of a on b = a.dot(b) / b.lengthSquared() * b
		Vec3f_Device __device__ projectOn(const Vec3f_Device &v) const {
			fp_t length		= projectLength(v);
			return Vec3f_Device(length * v[0], length * v[1], length * v[2]);
		}

		Vec3f_Device __device__ normalize() const {
			Vec3f_Device ret (*this);
			ret /= length();
			return ret;
		}

		fp_t __device__ length() const {
			return sqrt(lengthSquared());
		}

		fp_t __device__ lengthSquared() const {
			return n[0] * n[0] + n[1] * n[1] + n[2] * n[2];
		}
};


inline __device__ Vec3f_Device operator+(const Vec3f_Device &v1, const Vec3f_Device &v2) {
	return Vec3f_Device(v1[0] + v2[0], v1[1] + v2[1], v1[2] + v2[2]);
}


inline __device__ Vec3f_Device operator-(const Vec3f_Device &v1, const Vec3f_Device &v2) {
	return Vec3f_Device(v1[0] - v2[0], v1[1] - v2[1], v1[2] - v2[2]);
}


inline __device__ Vec3f_Device operator*(const Vec3f_Device &v1, const Vec3f_Device &v2) {
	return Vec3f_Device(v1[0] * v2[0], v1[1] * v2[1], v1[2] * v2[2]);
}


inline __device__ Vec3f_Device operator/(const Vec3f_Device &v1, const Vec3f_Device &v2) {
	return Vec3f_Device(v1[0] / v2[0], v1[1] / v2[1], v1[2] / v2[2]);
}


inline __device__ Vec3f_Device operator*(fp_t t, const Vec3f_Device &v) {
	return Vec3f_Device(t * v[0], t * v[1], t * v[2]);
}


inline __device__ Vec3f_Device operator/(Vec3f_Device v, fp_t t) {
	return Vec3f_Device(v[0] / t, v[1] /  t, v[2] / t);
}


inline __device__ Vec3f_Device operator*(const Vec3f_Device &v, fp_t t) {
	return Vec3f_Device(t * v[0], t * v[1], t * v[2]);
}


inline __device__ bool operator==(const Vec3f_Device &v1, const Vec3f_Device &v2) {
	return v1[0] == v2[0] && v1[1] == v2[1] && v1[2] == v2[2];
}


struct Ray_Device {
	Vec3f_Device	pos;
	Vec3f_Device	dir;

	__device__ Vec3f_Device getPosition() const {
		return pos;
	}

	__device__ Vec3f_Device getDirection() const {
		return dir;
	}
};


class Hitable_Sphere_Device {
	// Data
	public:
		fp_t	radius;
		Vec3f_Device	center;

	// Operation
	public:
		__device__ bool intersect(Vec3f_Device &normal, fp_t *distance, const Ray_Device *ray, fp_t t_min, fp_t t_max) const {
			Vec3f_Device	oc	= ray->pos - center;

			fp_t	a	= ray->dir.dot(ray->dir);
			fp_t	b	= oc.dot(ray->dir);
			fp_t	c	= oc.dot(oc) - radius * radius;

			fp_t	discriminant	= b * b - a * c;

			// no intersection
			if (discriminant <= 0) return false;

			// find the length of the ray
			// check if the ray is hit within the range
			fp_t	ray_length;

			ray_length = (-b - sqrt(b * b - a * c)) / a;
			if (ray_length < t_max && ray_length > t_min) goto RAY_HIT;

			ray_length = (-b + sqrt(b * b - a * c)) / a;
			if (ray_length < t_max && ray_length > t_min) goto RAY_HIT;

			return false;

			// ray hit the object within the range
			// need to set the content of hit record
			RAY_HIT:
			*distance	= ray_length;
			normal		= (ray->pos + ray_length * ray->dir).normalize();

			return true;
 		}
};


class Hitable_Sphere {
	// Data
	public:
		fp_t			radius;
		Vec3f			center;
};


struct RecordRender {
	int		size_hitable;
	int		width;
	int		height;
};


// CUDA Function
__device__ float get_random(unsigned int *seed0, unsigned int *seed1) {
	*seed0 = 36969 * ((*seed0) & 65535) + ((*seed0) >> 16);  // hash the seeds using bitwise AND and bitshifts
	*seed1 = 18000 * ((*seed1) & 65535) + ((*seed1) >> 16);
   
	unsigned int ires = ((*seed0) << 16) + (*seed1);
   
	// convert to float
	union {
		float			f;
		unsigned int	ui;
	} res;
   
	res.ui = (ires & 0x007fffff) | 0x40000000;  // bitwise AND, bitwise OR
   
	return (res.f - 2.f) / 2.f;
}


__device__ bool intersect_scene(const Ray_Device *ray, Hitable_Sphere_Device *scene, int size_hitable) {
	for (int i = 0; i < size_hitable; ++i) {
		
		Vec3f_Device	normal;
		fp_t	distance;
		
		// TODO: select a better t_max
		if (!scene[i].intersect(normal, &distance, ray, 0, 8192.0f)) continue;
		return true;
	}

	return false;
}


// __global__: executed on the device (GPU) and callable onbly from host (CPU)
// this kernel runs in parallel on all the CUDA threads
__global__ void render_kernel(Vec3f_Device *output, Hitable_Sphere_Device *scene, RecordRender *record_render) {

	// assign a CUDA thread to every pixel (x, y)
	// blockIdx, blockDim, threadIdex are CUDA specific keywords
	// replaces nested outer loops in CPU code looping over image rows and image columns
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// int i = (record_render->height - y - 1) * record_render->width + x;  // index of current pixel (calculated using thread index)
	int i = y * record_render->width + x;

	// unsigned int s1 = x;  // seeds for random number generator
	// unsigned int s2 = y;

	// generate ray directed at lower left corner of the screen
	// compute direction of all other rays by adding cx and cy increments in x and y direction
	Ray_Device ray;
	ray.pos = Vec3f_Device(0.0f, 0.0f, 0.0f);
	// ray.dir	= Vec3f_Device(
	// 	0.0f + ((float)x - (record_render->width / 2)) / (record_render->width), 
	// 	0.0f + ((float)y - (record_render->height / 2)) / (record_render->height), 
	// 	1.0f);

	Vec3f_Device result;

	for (int i = 0; i < 8; ++i) {
		for (int j = 0; j < 8; ++j) {
			ray.dir	= Vec3f_Device(
				0.0f + ((float)x - (record_render->width / 2) + ((float)i - 4.0) / 8) / (record_render->width), 
				0.0f + ((float)y - (record_render->height / 2) + ((float)j - 4.0) / 8) / (record_render->height), 
				1.0f);

			result += intersect_scene(&ray, scene, record_render->size_hitable) ? Vec3f_Device(1.0 / 64.0, 1.0 / 64.0, 1.0 / 64.0) : Vec3f_Device();
		}
	}

	output[i] = result;
}


int main(void) {
	// MARK: message
	printf("Main started \n");

	const int width		= 128;
	const int height	= 128;

	// data - output
	printf("Data: Output \n");

	Vec3f			*output_host;
	Vec3f_Device	*output_device;

	output_host	= new Vec3f[width * height];
	hipMalloc(&output_device, width * height * sizeof(Vec3f_Device));

	// data - scene
	printf("Data: Scene \n");
	
	Hitable_Sphere 			*hitable_host;
	Hitable_Sphere_Device	*hitable_device;

	hitable_host = new Hitable_Sphere[1];
	hitable_host[0].radius = 0.5;
	hitable_host[0].center = Vec3f(0, 0, 2);

	hipMalloc(&hitable_device, 1 * sizeof(Hitable_Sphere_Device));
	hipMemcpy(hitable_device, hitable_host, 1 * sizeof(Hitable_Sphere_Device), hipMemcpyHostToDevice);

	// data - render
	printf("Data: Render \n");

	RecordRender record_render_host;
	RecordRender *record_render_device;

	record_render_host.size_hitable	= 1;
	record_render_host.width		= width;
	record_render_host.height		= height;
	
	hipMalloc(&record_render_device, sizeof(RecordRender));
	hipMemcpy(record_render_device, &record_render_host, sizeof(RecordRender), hipMemcpyHostToDevice);

	// dim3 is CUDA specific type, block and grid are required to schedule CUDA threads over streaming multiprocessors
	dim3	threads	= dim3(8, 8);
	dim3	blocks	= dim3(width / threads.x, height / threads.y);
	
	// MARK: messaging
	printf("CUDA initiaized \n");
	printf("Start rendering... \n");

	// record time
	float gpu_time = 0.0f;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// StopWatchInterface *timer = NULL;
	// sdkCreateTimer(&timer);
	// sdkResetTimer(&timer);
	// checkCudaErrors(hipDeviceSynchronize());

	// schedule threads on device and launch CUDA kernel from host
	// then copy result of computation from device back to host
	// sdkStartTimer(&timer);
	hipEventRecord(start);

	render_kernel<<<blocks, threads>>>(output_device, hitable_device, record_render_device);
	checkCudaErrors(hipMemcpy(output_host, output_device, width * height * sizeof(Vec3f_Device), hipMemcpyDeviceToHost));
	
	// sdkStopTimer(&timer);
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// show time
	checkCudaErrors(hipEventElapsedTime(&gpu_time, start, stop));
	printf("Time elapsed: %f \n", gpu_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	// free CUDA memory
	checkCudaErrors(hipFree(output_device));
	checkCudaErrors(hipFree(hitable_device));
	checkCudaErrors(hipFree(record_render_device));

	// MARK: messaging
	printf("Done\n");

	// write image to PPM file
	FILE *f = fopen("result.ppm", "w");
	fprintf(f, "P3\n%d %d\n%d\n", width, height, 255);

	// loop over pixels, write RGB
	for (int i = 0; i < width * height; ++i) {
		fprintf(f, "%d %d %d \n", 
			(int)(output_host[i][0] * 255),
			(int)(output_host[i][1] * 255),
			(int)(output_host[i][2] * 255));
	}

	printf("Saved image to 'result.ppm'\n");

	delete[] output_host;
	delete[] hitable_host;
	printf("Done \n");
	
	return 0;
}

#include "../inc/RayTracer_Texture_Constant.cuh"
#include "../inc/RayTracer_Material.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Material::Material() {
    transmissive    = nullptr;
    index           = RAY_INDEX_AIR;
}


__device__ Material::~Material() {
}


__device__ error_t Material::allocateScatter(int32_t size) {
    // free old
    hipFree(scatter_list);

    // allocate new
    hipMalloc(&scatter_list, size * sizeof(Scatter*));
    scatter_index   = 0;
    scatter_size    = size;

    return ERROR_NO;
}


// TODO: missing uniquness check
__device__ error_t Material::addScatter(Scatter *scatter) {
    if (scatter_index == scatter_size) return ERROR_ANY;

    scatter_list[scatter_index] = scatter;
    scatter_index++;
    return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Material::rmScatter(Scatter *scatter) {
    return ERROR_ANY;
}

// Static Function Implementation
// ...

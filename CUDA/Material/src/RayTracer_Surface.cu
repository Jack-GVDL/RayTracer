#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Surface.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// TODO: rename
__global__ static void	setTextureImage_data	(Texture_Image *texture, Vec3f *data, int32_t w, int32_t h);


// Operation Handling
__host__ bool Surface::convertToTexture(Texture_Image *texture) {
	// check if data and texture exist or not
	if (data == nullptr)	return false;
	if (texture == nullptr)	return false;
	
	// TODO: currently can only handle RGB_888
	if (bit_pixel != 24)	return false;

	// malloc for temp space
	Vec3f *color_host;
	Vec3f *color_device;

	color_host = new Vec3f[width * height];
	hipMalloc(&color_device, width * height * sizeof(Vec3f));

	// texture use RGB_888
	fp_t			color_r, color_g, color_b;
	const int32_t	offset_pixel	= (bit_pixel + 7) / 8;
	const int32_t	offset_r		= 0;
	const int32_t	offset_g		= 1;
	const int32_t	offset_b		= 2;

	for (int i = 0; i < width * height; i++) {
		// get RGB and convert it double
		// where the range is [0, 1] (0, 1 is inclusive)
		// TODO: currently assume the range of a single channel is [0, 255]
		color_r = (fp_t)(data[i * offset_pixel + offset_r]) / 255.0;
		color_g = (fp_t)(data[i * offset_pixel + offset_g]) / 255.0;
		color_b = (fp_t)(data[i * offset_pixel + offset_b]) / 255.0;

		color_host[i] = Vec3f(color_r, color_g, color_b);
	}

	// memcpy to device and call kernel
	hipMemcpy(color_device, color_host, width * height * sizeof(Vec3f), hipMemcpyHostToDevice);
	setTextureImage_data <<< 1, 1 >>> (texture, color_device, width, height);

	// free for temp space
	delete[] color_host;

	return true;
}


// Static Function Implementation
__global__ static void setTextureImage_data(Texture_Image *texture, Vec3f *data, int32_t w, int32_t h) {
	texture->color	= data;
	texture->width	= w;
	texture->height	= h;
}

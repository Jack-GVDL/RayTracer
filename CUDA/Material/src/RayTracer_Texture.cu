#include "../inc/RayTracer_Texture.cuh"


// Define
// TODO: may use stack buffer in future
#define MAX_LENGTH_VEC_BUFFER	8


// Static Data
// ...


// Static Function Prototype
__device__ static void	get_pixel	(Vec3f &dst, const Texture *texture, const Vec3f &src);


// Operation Handling
__device__ Texture::Texture() {
}


__device__ Texture::~Texture() {
	hipFree(input_list);
}


__device__ error_t Texture::addInput(Texture *texture, int32_t index) {
	if (index < 0 || index >= input_size) return ERROR_ANY;
	input_list[index]	= texture;
	return ERROR_NO;
}


__device__ error_t Texture::rmInput(int32_t index) {
	if (index < 0 || index >= input_size) return false;
	input_list[index]	= nullptr;
	return true;
}


__device__ void Texture::getPixel(Vec3f &dst, const Vec3f &src) const {
	get_pixel(dst, this, src);
}


// Static Function Implementation
__device__ static void get_pixel(Vec3f &dst, const Texture *texture, const Vec3f &src) {
	// backtracking
	Vec3f point_list[MAX_LENGTH_VEC_BUFFER];
	for (int32_t i = 0; i < texture->input_size; i++) {

		if (texture->input_list[i] == nullptr) {
			point_list[i] = Vec3f(0);
			continue;
		}

		Vec3f temp;
		get_pixel(temp, texture->input_list[i], src);
		point_list[i] = temp;

	}

	// add src
	point_list[texture->input_size] = src;

	// local
	texture->_getPixel_(dst, point_list);
}

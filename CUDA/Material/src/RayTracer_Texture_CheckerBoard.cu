#include "hip/hip_runtime.h"
#include <cmath>
#include "../inc/RayTracer_Texture_CheckerBoard.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Texture_CheckerBoard::Texture_CheckerBoard() {
	input_size	= 0;
	input_list	= nullptr;
	for (int32_t i = 0; i < input_size; i++) input_list[i] = nullptr;
}


__device__ Texture_CheckerBoard::~Texture_CheckerBoard() {
}


__device__ void Texture_CheckerBoard::setBoardSize(const Vec3f &size) {
	size_board = size;
}


__device__ void Texture_CheckerBoard::setBoardColor(const Vec3f &color_1, const Vec3f &color_2) {
	this->color_1	= color_1;
	this->color_2	= color_2;
}


// pixel cannot be set in this way
// so do nothing
__device__ void Texture_CheckerBoard::setPixel(const Vec3f &point, const Vec3f &pixel) {
}


__device__ void Texture_CheckerBoard::_getPixel_(Vec3f &dst, Vec3f *src) const {
	// check if point should be black or white
	// one characteristic of sin / cos is periodic changes of y ranging [1, -1]
	fp_t sines 	= sin(size_board[0] * src[0][0]) * 
				  sin(size_board[1] * src[0][1]) * 
				  sin(size_board[2] * src[0][2]);

	if (sines < 0)	dst = color_1;
	else			dst = color_2;
}


// Static Function Implementation
// ...

#include "../inc/RayTracer_Texture_Image.cuh"

// TODO: test
#include <stdio.h>


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Texture_Image::Texture_Image() {
	input_size	= 1;
	// input_list	= new Texture*[input_size];
	hipMalloc(&input_list, input_size * sizeof(Texture*));
	for (int32_t i = 0; i < input_size; ++i) input_list[i] = nullptr;
}


__device__ Texture_Image::~Texture_Image() {
}


__device__ void Texture_Image::setPixel(const Vec3f &point, const Vec3f &pixel) {
}


__device__ void Texture_Image::_getPixel_(Vec3f &dst, Vec3f *src) const {
	// get point
	const int32_t	x = (int32_t)(src[0][0]);
	const int32_t	y = (int32_t)(src[0][1]);

	// check if out of bound
	if (x < 0 || x >= width || y < 0 || y >= height) {
		dst = Vec3f();
		return;
	}

	// get image pixel
	dst = color[x + y * width];
}


// Static Function Implementation
// ...

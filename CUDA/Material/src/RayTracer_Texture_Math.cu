#include "../inc/RayTracer_Texture_Math.cuh"

// TODO: test
#include <stdio.h>


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// ...


// Static Data
// ...


// Operation Handling
// additor
__device__ Texture_Math_Additor::Texture_Math_Additor() {
	input_size	= 1;
	// input_list	= new Texture*[input_size];
	hipMalloc(&input_list, input_size * sizeof(Texture*));
	for (int32_t i = 0; i < input_size; i++) input_list[i] = nullptr;
}


__device__ Texture_Math_Additor::~Texture_Math_Additor() {
}


__device__ void Texture_Math_Additor::setAdditor(const Vec3f &value) {
	this->additor = value;
}


__device__ void Texture_Math_Additor::setPixel(const Vec3f &point, const Vec3f &pixel) {
}


__device__ void Texture_Math_Additor::_getPixel_(Vec3f &dst, Vec3f *src) const {
	dst = src[0] + additor;
}


// multiplier
__device__ Texture_Math_Multiplier::Texture_Math_Multiplier() {
	input_size	= 1;
	// input_list	= new Texture*[input_size];
	hipMalloc(&input_list, input_size * sizeof(Texture*));
	for (int32_t i = 0; i < input_size; i++) input_list[i] = nullptr;
}


__device__ Texture_Math_Multiplier::~Texture_Math_Multiplier() {
}


__device__ void Texture_Math_Multiplier::setMultiplier(const Vec3f &value) {
	this->multiplier = value;
}


__device__ void Texture_Math_Multiplier::setPixel(const Vec3f &point, const Vec3f &pixel) {
}


__device__ void Texture_Math_Multiplier::_getPixel_(Vec3f &dst, Vec3f *src) const {
	dst = src[0] * multiplier;
}


// Static Function Implementation
// ...

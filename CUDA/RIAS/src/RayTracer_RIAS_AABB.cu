#include "hip/hip_runtime.h"
#include "../inc/RayTracer_RIAS_AABB.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
static Hitable_AABB*	*buffer_hitable	= nullptr;
static AABB*			*buffer_aabb	= nullptr;

static compare_func_t	compare_func_list[] = {
	compare_x_axis,
	compare_y_axis,
	compare_z_axis
};


// Static Function Prototype
// host
__host__ static void			host_createHitable			(Hitable_AABB* *dst);
__host__ static void			host_createAABB				(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right);
__host__ static inline void		host_getBounding			(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size);
__host__ static void			host_buildTree				(AABB* *dst, RecordBounding *record_list, int32_t size);

// global
__global__ static void			global_createHitable		(Hitable_AABB* *dst);
__global__ static void			global_createAABB			(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right);
__global__ static void			global_getBounding			(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size, int32_t offset);
__global__ static void			global_setRoot				(Hitable_AABB *hitable_aabb, AABB *aabb);

// compare function
__host__ static inline int32_t	select_compare_func			(RecordBounding *record_list, int32_t size);
__host__ static fp_t			get_diff					(RecordBounding *record_list, int32_t size, int32_t axis);
__host__ static int32_t			compare_x_axis				(const void *a, const void *b);
__host__ static int32_t			compare_y_axis				(const void *a, const void *b);
__host__ static int32_t			compare_z_axis				(const void *a, const void *b);

// device
// ...


// Operation Handling
__host__ RIAS_AABB() {
}


__host__ ~RIAS_AABB() {
}


// TODO: not yet corrected
__host__ virtual error_t RIAS_AABB::load(SceneObject_Hitable* *hitable_list, int32_t size) {
	// create hitable
	host_createHitable(&hitable);

	// TODO: not yet corrected
	// get bounding
	RecordBounding *record_list;
	record_list = (RecordBounding*)malloc(size * sizeof(RecordBounding));

	host_getBounding(record_list, hitable_list, size);

	// build tree
	// TODO: should global_aabb_setRoot be inside host_aabb_buildTree ?
	AABB *aabb_device;
	host_buildTree(&aabb_device, record_list, size);
	global_setRoot <<< 1, 1 >>> (hitable, aabb_device);

	// free space
	free(record_list);

	return ERROR_NO;
}


// Static Function Implementation
// host
__host__ static void host_createHitable(Hitable_AABB* *dst) {
	// check if need to allocate buffer
	if (buffer_hitable == nullptr) hipMalloc(&buffer_hitable, sizeof(Hitable_AABB*));

	// create hitable
	global_createHitable <<< 1, 1 >>> (buffer_hitable);

	// transfer this pointer from device to host
	hipMemcpy(dst, buffer_hitable, sizeof(Hitable_AABB*), hipMemcpyDeviceToHost);
}


__host__ static void host_createAABB(
	AABB* *dst, 
	AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right) {

	// check if need to allocate buffer
	if (buffer_aabb == nullptr) hipMalloc(&buffer_aabb, sizeof(AABB*));

	// create aabb
	global_createAABB <<< 1, 1 >>> (buffer_aabb, aabb_left, aabb_right, hitable_left, hitable_right);

	// transfer this pointer from device to host
	hipMemcpy(dst, buffer_aabb, sizeof(AABB*), hipMemcpyDeviceToHost);
}


__host__ static inline void host_getBounding(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size) {
	RecordBounding			*record_list_device;
	SceneObject_Hitable*	*hitable_list_device;

	// allocate device space
	hipMalloc(&record_list_device, size * sizeof(RecordBounding));
	hipMalloc(&hitable_list_device, size * sizeof(SceneObject_Hitable*));
	
	// transfer hitable_list from host to device
	hipMemcpy(hitable_list_device, hitable_list, size * sizeof(SceneObject_Hitable*), hipMemcpyHostToDevice);

	// TODO: the number of thread should be dynamic
	global_getBounding <<< 1, 10 >>> (record_list_device, hitable_list_device, size, 10);

	// transfer record_list from device to host
	hipMemcpy(record_list, record_list_device, size * sizeof(RecordBounding), hipMemcpyDeviceToHost);

	// free device space
	hipFree(record_list_device);
	hipFree(hitable_list_device);
}


__host__ static void host_buildTree(AABB* *dst, RecordBounding *record_list, int32_t size) {	
	// base case
	// when size <= 2, then this aabb node is a leaf
	if (size == 0) {
		host_createAABB(dst, nullptr, nullptr, nullptr, nullptr);
		return;
	}

	if (size == 1) {
		host_createAABB(dst, nullptr, nullptr, record_list[0].hitable, nullptr);
		return;
	}

	if (size == 2) {
		host_createAABB(dst, nullptr, nullptr, record_list[0].hitable, record_list[1].hitable);
		return;
	}

	// recursive case
	// sort the list
	int32_t index_compare = select_compare_func(record_list, size);
	qsort(record_list, size, sizeof(RecordBounding), compare_func_list[index_compare]);

	// build the tree recursively
	AABB *left, *right;
	host_buildTree(&left, record_list, size / 2);
	host_buildTree(&right, record_list + size / 2, size - size / 2);
	
	// build this level of aabb
	host_createAABB(dst, left, right, nullptr, nullptr);
}


// global
__global__ static void global_createHitable(Hitable_AABB* *dst) {
	*dst = new Hitable_AABB();
}


__global__ static void global_createAABB(AABB* *dst, AABB *aabb_left, AABB *aabb_right, SceneObject_Hitable *hitable_left, SceneObject_Hitable *hitable_right) {
	*dst = new AABB(aabb_left, aabb_right, hitable_left, hitable_right);
}


__global__ static void global_getBounding(RecordBounding *record_list, SceneObject_Hitable* *hitable_list, int32_t size, int32_t offset) {
	// parallle processing
	int32_t global_index = blockIdx.x * blockDim.x + threadIdx.x; 

	// just copying (copy constructor)
	for (int32_t i = 0; i < size; i += offset) {
		if (i + global_index >= size) continue;

		record_list[i + global_index].bounding		= hitable_list[i + global_index]->bounding;
		record_list[i + global_index].hitable		= hitable_list[i + global_index];
	}
}


__global__ static void global_setRoot(Hitable_AABB *hitable_aabb, AABB *aabb) {
	hitable_aabb->setAABB(aabb);
}


// device
// ...


// compare function
__host__ static inline int32_t select_compare_func(RecordBounding *record_list, int32_t size) {
	/* The purpose of this function is to find out in which axis
	   the sorting will result the best / better for tree building

	   Currently, it is achieved by find the diff between min and max of the value of
	   midpoint of bounding box in each axis
	   The larger the differences for the axis
	   the larger the possibility that the sort in this axis will be better
	 */

	 // find the difference of min and max
	 fp_t diff_x = get_diff(record_list, size, 0);
	 fp_t diff_y = get_diff(record_list, size, 1);
	 fp_t diff_z = get_diff(record_list, size, 2);

	 // return the axis index
	 return diff_x > diff_y && diff_x > diff_z ? 0 : diff_y > diff_z ? 1 : 2;
}


__host__ static fp_t get_diff(RecordBounding *record_list, int32_t size, int32_t axis) {
	if (size == 0) return 0;

	fp_t min_, max_, mid;
	mid = 
		record_list[0].bounding.min_[axis] + 
		(record_list[0].bounding.max_[axis] - record_list[0].bounding.min_[axis]) / 2;
	
	min_ = mid;
	max_ = mid;

	for (int32_t i = 1; i < size; ++i) {
		mid = 
			record_list[i].bounding.min_[axis] + 
			(record_list[i].bounding.max_[axis] - record_list[i].bounding.min_[axis]) / 2;

		min_ = std::min<fp_t>(mid, min_);
		max_ = std::max<fp_t>(mid, max_);
	}

	// return std::abs<int32_t>(max_ - min_);
	return max_ - min_;
}


__host__ static int32_t compare_x_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[0] + bounding_1->bounding.max_[0];
	int32_t	mid_2	= bounding_2->bounding.min_[0] + bounding_2->bounding.max_[0];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}


__host__ static int32_t compare_y_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[1] + bounding_1->bounding.max_[1];
	int32_t	mid_2	= bounding_2->bounding.min_[1] + bounding_2->bounding.max_[1];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}


__host__ static int32_t compare_z_axis(const void *a, const void *b) {
	RecordBounding *bounding_1 = (RecordBounding*)a;
	RecordBounding *bounding_2 = (RecordBounding*)b;

	int32_t	mid_1	= bounding_1->bounding.min_[2] + bounding_1->bounding.max_[2];
	int32_t	mid_2	= bounding_2->bounding.min_[2] + bounding_2->bounding.max_[2];

	// compare by mid-point32_t of hitable
	if (mid_1 < mid_2)	return -1;
	if (mid_1 == mid_2)	return 0;
	return 1;
}

#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Scatter.hpp"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
__device__ static inline void*	get_record						(int32_t index, void *memory, int32_t offset);

// backup
/*
__device__ static inline void	schedule_check_collision		(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back);
__device__ static inline void	schedule_load_scatter			(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back);
__device__ static inline void	schedule_execute_scatter		(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back);
__device__ static inline void	schedule_accumulate_intensity	(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back);
*/


// Operation Handling
// memory control
__device__ MemoryControl_Scatter::MemoryControl_Scatter() {
	// offset must be power of 2 and divisible by 4KB
	// offset must larger than sizeof RecordRay
	int32_t size_record	= (int32_t)(sizeof(RecordRay));
	offset				= 1;

	for (uint8_t i = 0; i < 8; i++) {

		// offset must be power of 2 and larger than sizeof RecordRay
		if (offset < size_record) {
			offset *= 2;
			continue;
		}

		break;
	}
}


__device__ MemoryControl_Scatter::~MemoryControl_Scatter() {
}


__device__ void MemoryControl_Scatter::setMemory(void *memory, int32_t size) {
	// when take over another memory space,
	// need to reset the queue
	record_list		= memory;
	record_size		= size / offset;

	// reset memory space
	reset();
}


__device__ void MemoryControl_Scatter::reset() {
	index_empty = 0;
}


__device__ void* MemoryControl_Scatter::createRecord() {
	if (index_empty == -1 || index_empty == record_size) return nullptr;

	// get an empty record
	RecordRay *record = (RecordRay*)get_record(index_empty, record_list, offset);
	index_empty++;

	// no need to clear record content
	// bzero(record, sizeof(RecordRay));

	return record;
}


// backup
// it is assume that record must be in ready queue
// __device__ void MemoryControl_Scatter::waitRecord(RecordRay *record) {
// }
// 
// 
// __device__ void MemoryControl_Scatter::destroyRecord(RecordRay *record) {
// }


__device__ void* MemoryControl_Scatter::getRecord(int index) {
	return get_record(index, record_list, offset);
}


// scatter
__device__ Scatter::Scatter() {
}


__device__ Scatter::~Scatter() {
	delete[] texture_list;
}


// TODO: missing: CUDA operation check (checkCudaErrors)
__device__ error_t Scatter::allocateScatter(int size) {
	// free the original memory space
	hipFree(scatter_list);

	// allocate new space
	hipMalloc(&scatter_list, size * sizeof(Scatter*));
	scatter_size	= size;
	scatter_index	= 0;

	return ERROR_NO;
}


// TODO: missing: uniqueness check
__device__ error_t Scatter::addScatter(Scatter *scatter) {
	if (scatter_index == scatter_size) return ERROR_ANY;
	scatter_list[scatter_index] = scatter;
	scatter_index++;
	return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Scatter::rmScatter(Scatter *scatter) {
	return ERROR_ANY;
}


__device__ error_t Scatter::setTexture(Texture *texture, int offset) {
	if (texture_list == nullptr)				return false;
	if (offset < 0 || offset >= texture_size)	return false;
	
	texture_list[offset] = texture;
	return true;
}


__device__ Texture* Scatter::getTexture(int offset) {
	if (offset < 0 || offset >= texture_size) return nullptr;
	return texture_list[offset];
}


__device__ void Scatter::setRecord_tree(RecordRay *dst, RecordRay *src) const {
	dst->parent					= src;
	dst->scene					= src->scene;
	dst->outer					= src->outer;
	dst->depth					= src->depth - 1;

	dst->threshold				= Vec3f(1);
	dst->intensity				= Vec3f(0);

	dst->is_enable_hit			= 1;
	dst->scatter_source			= 1;

	dst->record_hit.length_min	= RAY_EPSILON;
	dst->record_hit.length_max	= std::numeric_limits<fp_t>::max();
}


__device__ void Scatter::setRecord_ray(RecordRay *dst, RecordRay *src, const Ray &ray) const {
	dst->record_hit.record.ray = ray;
}


__device__ void Scatter::setRecord_threshold(RecordRay *dst, RecordRay *src, const Vec3f &ratio) const {
	dst->threshold	= src->threshold.prod(ratio);
	src->threshold	-= dst->threshold;
}


__device__ void Scatter::setRecord_scatter(RecordRay *dst, RecordRay *src) const {
	if (scatter_index == 0) {
		dst->scatter_source = 1;
		return;
	}

	dst->scatter_source					= 0;
	dst->record_scatter.scatter_list	= scatter_list;
	dst->record_scatter.size			= scatter_index;
}


// scheduler
__device__ Scheduler_Scatter::Scheduler_Scatter() {
}


__device__ Scheduler_Scatter::~Scheduler_Scatter() {
}


__device__ void Scheduler_Scatter::setRoot(RecordRay *record) {
	memory_control.reset();
	queue = 0;

	// create record
	RecordRay *temp = (RecordRay*)memory_control.createRecord();
	if (temp == nullptr) return;

	// copy from other source
	hipMemcpy(temp, record, sizeof(RecordRay));
}


__device__ void Scheduler_Scatter::getRoot(RecordRay *record) {
	RecordRay *temp = (RecordRay*)memory_control.getRecord(0);
	if (temp == nullptr) return;

	// need to normalize resultant intensity
	// it should be ranging between 0 and 1
	temp->intensity = temp->intensity.clamp(0, 1);
	hipMemcpy(record, temp, sizeof(RecordRay));
}


__device__ void Scheduler_Scatter::setScene(Scene *scene) {
	this->scene = scene;
}


// currently no parallel processing
// so all the stuff will be done here
//
// return 0 if no record is queuing, else 1
// TODO: function too long, split the function, use static inline
__device__ int8_t Scheduler_Scatter::schedule() {
	// read ray queue
	// quit if no record is queuing
	if (queue == memory_control.index_empty) return 0;
	int32_t index_empty = memory_control.index_empty;
	
	RecordRay		*top		= (RecordRay*)memory_control.getRecord(0);
	RecordRay		*record;
	const Shader	*shader;

	// collision check
	// currently no parallel processing
	// backup
	// schedule_check_collision(		scene, &memory_control, top, queue, index_empty	);
	// schedule_load_scatter(			scene, &memory_control, top, queue, index_empty	);
	// schedule_execute_scatter(		scene, &memory_control, top, queue, index_empty	);
	// schedule_accumulate_intensity(	scene, &memory_control, top, queue, index_empty	);

	for (int32_t i = queue; i < index_empty; i++) {
		record = (RecordRay*)memory_control.getRecord(i);
	
		if (record->depth == 0)	{
			continue;
		}
		if (!record->is_enable_hit)	{
			record->is_hit = false;
			continue;
		}
	
		record->is_hit = scene->hit(&(record->record_hit));
	}

	// load scatter
	for (int32_t i = queue; i < index_empty; i++) {
		record = (RecordRay*)memory_control.getRecord(i);
		if (record->depth == 0) continue;
		
		switch (record->scatter_source) {
			// 0: already in record
			case 0:
				break;

			// 1: hit scene object, else NULL
			case 1:
				SceneObject_Hitable *object = record->record_hit.record.object;
				if (record->is_hit)	{
					record->record_scatter.scatter_list = object->shader.scatter_list.data();
					record->record_scatter.size			= object->shader.scatter_list.size();
					record->record_scatter.index		= 0;

				} else {
					record->record_scatter.scatter_list = nullptr;
					record->record_scatter.size			= 0;
					record->record_scatter.index		= 0;

				}
				break;
		}

	}

	// scatter operation
	for (int32_t i = queue; i < index_empty; i++) {
		record = (RecordRay*)memory_control.getRecord(i);
		if (record->depth == 0) 							continue;
		if (record->record_scatter.scatter_list == nullptr)	continue;

		// scatter operation
		// TODO: future
		// record = (RecordRay*)memory_control.getRecord(i);
		// record->record_scatter.scatter_list[record->record_scatter.index]->scatter(record, &memory_control);

		for (int16_t i = 0; i < record->record_scatter.size; i++) {
			record->record_scatter.scatter_list[i]->scatter(record, &memory_control);
		}
	}

	// intensity accumulation
	for (int32_t i = queue; i < index_empty; i++) {
		record = (RecordRay*)memory_control.getRecord(i);
		if (record->depth == 0) continue;

		top->intensity += record->intensity;
	}

	// configure to get ready to next scheduling
	queue = index_empty;

	if (top->intensity[0] > 1.0 && top->intensity[1] > 1.0 && top->intensity[2] > 1.0) return 0;
	return 1;
}


// Static Function Implementation
__device__ static inline void* get_record(int32_t index, void *memory, int32_t offset) {
	return (uint8_t*)memory + index * offset;
}


// backup
/*
__device__ static inline void schedule_check_collision(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back) {
	RecordRay *record;

	for (int32_t i = front; i < back; i++) {
		record = (RecordRay*)control->getRecord(i);

		// skip record
		if (record->depth == 0)	{
			continue;
		}
		if (!record->is_enable_hit)	{
			record->is_hit = false;
			continue;
		}

		record->is_hit = scene->hit(&(record->record_hit));
	}
}


__device__ static inline void schedule_load_scatter(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back) {
	RecordRay *record;
	
	for (int32_t i = front; i < back; i++) {
		record = (RecordRay*)control->getRecord(i);

		// skip record
		if (record->depth == 0) continue;
		
		switch (record->scatter_source) {
			// 0: already in record
			case 0:
				break;

			// 1: hit scene object, else NULL
			case 1:
				SceneObject_Hitable *object = record->record_hit.record.object;
				if (record->is_hit)	{
					record->record_scatter.scatter_list = object->shader.scatter_list.data();
					record->record_scatter.size			= object->shader.scatter_list.size();
					record->record_scatter.index		= 0;

				} else {
					record->record_scatter.scatter_list = nullptr;
					record->record_scatter.size			= 0;
					record->record_scatter.index		= 0;

				}
				break;
		}

	}
}


__device__ static inline void schedule_execute_scatter(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back) {
	RecordRay *record;
	
	for (int32_t i = front; i < back; i++) {
		record = (RecordRay*)control->getRecord(i);

		// skip record
		if (record->depth == 0) 							continue;
		if (record->record_scatter.scatter_list == nullptr)	continue;

		// scatter operation
		// TODO: future
		// record = (RecordRay*)memory_control.getRecord(i);
		// record->record_scatter.scatter_list[record->record_scatter.index]->scatter(record, &memory_control);

		for (int16_t i = 0; i < record->record_scatter.size; i++) {
			record->record_scatter.scatter_list[i]->scatter(record, control);
		}
	}
}


__device__ static inline void schedule_accumulate_intensity(Scene *scene, MemoryControl_Scatter *control, RecordRay *top, int32_t front, int32_t back) {
	RecordRay *record;
	
	for (int32_t i = front; i < back; i++) {
		record = (RecordRay*)control->getRecord(i);
		if (record->depth == 0) continue;

		top->intensity += record->intensity;
	}
}
*/

#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Camera.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Camera::Camera() {
	this->look_from		= Vec3f(0.0, 0.0, 0.0);
	this->look_at		= Vec3f(0.0, 0.0, 1.0);
	this->up			= Vec3f(0.0, 1.0, 0.0);
	this->fov			= 90;
	this->aspect		= 1;

	update();
}


__device__ Camera::Camera(const Vec3f look_from, const Vec3f look_at, const Vec3f up, fp_t fov, fp_t aspect) {
	this->look_from	= look_from;
	this->look_at	= look_at;
	this->up		= up;
	this->fov		= fov;
	this->aspect	= aspect;

	update();
}


__device__ void Camera::setLookFrom(const Vec3f look_from) {
	this->look_from = look_from;
	update();
}


__device__ void Camera::setLookAt(const Vec3f look_at) {
	this->look_at	= look_at;
	update();
}


__device__ void Camera::setUpDirection(const Vec3f up) {
	this->up		= up;
	update();
}


__device__ void Camera::setFOV(fp_t value) {
	this->fov = value;
	update();
}


__device__ void Camera::setAspectRatio(fp_t value) {
	this->aspect = value;
	update();
}


__device__ void Camera::setAll(const Vec3f look_from, const Vec3f look_at, const Vec3f up, fp_t fov, fp_t aspect) {
	this->look_from	= look_from;
	this->look_at	= look_at;
	this->up		= up;
	this->fov		= fov;
	this->aspect	= aspect;

	update();
}


__device__ void Camera::update() {
	// fov
	// TODO: need more explanation on FOV
	//
	// aspect ratio: ratio of is width to its height
	// aspect = width / height
	// height = width / aspect
	// width  = aspect / height
	fp_t	theta		= fov * M_PI / 180;
	fp_t	height_half	= tan(theta / 2);
	fp_t	width_half	= aspect / height_half;

	// backup
	// width	= width_half;
	// height	= height_half;

	// w: direction
	// u: x-axis
	// v: y-axis
	// corss product of two normalized vectors should also be a normalized vector
	w	= (look_at - look_from).normalize();
	u	= w.cross(up).normalize();
	v	= u.cross(w);

	u	*= width_half;
	v	*= height_half; 
}


__device__ Ray Camera::getRay(fp_t x, fp_t y) const {
	return Ray(look_from, (w + x * u + y * v).normalize());
}


// Static Function Implementation
// ...

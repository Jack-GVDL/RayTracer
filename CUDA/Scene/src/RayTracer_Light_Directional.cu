#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Light_Directional.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// ...


// Static Data
// ...


// Operation Handling
__device__ SceneObject_Light_Directional::SceneObject_Light_Directional() {
}


__device__ SceneObject_Light_Directional::~SceneObject_Light_Directional() {
}


__device__ void SceneObject_Light_Directional::setOrientation(const Vec3f &orientation) {
	this->orientation = orientation;
}


__device__ fp_t SceneObject_Light_Directional::getAttenuation(const Vec3f &point) const {
	return 1.0;
}


__device__ fp_t SceneObject_Light_Directional::getDistance(const Vec3f &point) const {
	return std::numeric_limits<fp_t>::max();
}


__device__ Vec3f SceneObject_Light_Directional::getColor(const Vec3f &point) const {
	return color;
}


__device__ Vec3f SceneObject_Light_Directional::getDirection(const Vec3f &point) const {
	return orientation;
}


// Static Function Implementation
// ...

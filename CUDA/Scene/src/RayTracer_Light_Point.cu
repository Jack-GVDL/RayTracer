#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Light_Point.cuh"


// Define
// ...


// Typedef
// ...


// Static Function Prototype
// ...


// Static Data
// ...


// Operation Handling
__device__ Light_Point::Light_Point() {
}


__device__ Light_Point::~Light_Point() {
}


__device__ void Light_Point::setAttenuation(const Vec3f &coeff) {
	this->attenuation = coeff;
}


__device__ fp_t Light_Point::getAttenuation(const Vec3f &point) const {
	const fp_t		coeff_1	= attenuation[0];
	const fp_t		coeff_2	= attenuation[1];
	const fp_t		coeff_3	= attenuation[2];

	const fp_t		d2		= (point - origin).lengthSquared();
	const fp_t		d1		= sqrt(d2);
	const fp_t		result	= coeff_1 + coeff_2 * d1 + coeff_3 * d2;

	// do not divide by zero
	return result == 0 ? 1 : min<fp_t>(1 / result, 1);
}


__device__ fp_t Light_Point::getDistance(const Vec3f &point) const {
	return (origin - point).length();
}


__device__ Vec3f Light_Point::getColor(const Vec3f &point) const {
	return color;
}


__device__ Vec3f Light_Point::getDirection(const Vec3f &point) const {
	return (origin - point).normalize();
}


// Static Function Implementation
// ...

#include "../inc/RayTracer_Scene.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Scene::Scene() {
}


__device__ Scene::~Scene() {
}


__device__ error_t Scene::allocateHitable(int32_t size) {
	// free old
	hipFree(hitable_list);

	// allocate new
	hipMalloc(&hitable_list, size * sizeof(SceneObject_Hitable*));
	hitable_size	= size;
	hitable_index	= 0;

	return ERROR_NO;
}


// TODO: missing uniqueness checking
__device__ error_t Scene::addHitable(SceneObject_Hitable *hitable) {
	hitable_list[hitable_index] = hitable;
	hitable_index++;
	return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Scene::rmHitable(SceneObject_Hitable *hitable) {
	return ERROR_ANY;
}


__device__ error_t Scene::allocateLight(int32_t size) {
	// free old
	hipFree(light_list);

	// allocate new
	hipMalloc(&light_list, size * sizeof(SceneObject_Light*));
	light_size	= size;
	light_index	= 0;

	return ERROR_NO;
}


// TODO: missing uniqueness checking
__device__ error_t Scene::addLight(SceneObject_Light *light) {
	light_list[light_index] = light;
	light_index++;
	return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Scene::rmLight(SceneObject_Light *light) {
	return ERROR_ANY;
}


__device__ error_t Scene::allocateAmbient(int32_t size) {
	// free old
	hipFree(ambient_list);

	// allocate new
	hipMalloc(&ambient_list, size * sizeof(SceneObject_Light*));
	ambient_size	= size;
	ambient_index	= 0;

	return ERROR_NO;
}


// TODO: missing uniqueness checking
__device__ error_t Scene::addAmbient(SceneObject_Light *light) {
	ambient_list[ambient_index] = light;
	ambient_index++;
	return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Scene::rmAmbient(SceneObject_Light *light) {
	return ERROR_ANY;
}


__device__ Vec3f Scene::getAmbientIntensity() const {
	Vec3f result = Vec3f(0);

	for (int32_t i = 0; i < ambient_index; ++i) {
		SceneObject_Light *light = ambient_list[i];
		result += light->getColor(Vec3f());
	}

	return result;
}


__device__ int8_t Scene::hit(RecordHit_Extend *record) const {
	RecordHit	temp_record;
	int8_t		is_hit			= 0;
	fp_t		closest			= record->length_max;

	// set ray
	temp_record.ray = record->record.ray;

	// search for hitable in child
	for (int32_t i = 0; i < hitable_index; ++i) {
		SceneObject_Hitable *hitable = hitable_list[i];
		
		if (!hitable->hit(&temp_record, record->length_min, closest)) continue;
		// if (temp_record.distance > closest) continue;

		is_hit			= 1;
		closest			= temp_record.distance;
		record->record	= temp_record;
	}

	return is_hit;
}


// Static Function Implementation
// ...

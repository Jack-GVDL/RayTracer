#include "../inc/RayTracer_Sampler.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__host__ Sampler::Sampler():
buffer_dir			(nullptr),
buffer_image		(nullptr),
buffer_dir_size		(0),
buffer_image_size	(0),
w					(0),
h					(0),
block_w				(0),
block_h				(0)
{
}


__host__ Sampler::~Sampler() {
	hipFree(buffer_dir);
	hipFree(buffer_image);
}


__host__ error_t Sampler::setSizeImage(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;

	this->w = w;
	this->h = h;
	
	update();
	return ERROR_NO;
}


// Static Function Implementation
// ...

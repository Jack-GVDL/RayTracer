#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Sampler_Grid.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
__global__ void	convert_pre_mapper	(Vec3f *dst, Vec3f *src, int32_t src_w, int32_t src_h,
									 int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h);
__global__ void	convert_post_mapper	(Vec3f *dst, Vec3f *src, int32_t dst_w, int32_t dst_h, 
									 int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h);


// Operation Handling
__host__ Sampler_Grid::Sampler_Grid():
grid_w(0),
grid_h(0)
{
}


__host__ Sampler_Grid::~Sampler_Grid()  {
}


__host__ error_t Sampler_Grid::setSizeGrid(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;

	this->block_w	= w;
	this->block_h	= h;

	update();
	return ERROR_NO;
}


// backup
/*
__host__ error_t Sampler_Grid::setSizeImage(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;
	
	// set image size
	Sampler::setSizeImage(w, h);

	update();
	return ERROR_NO;
}
*/


// TODO: current assume that size is known and size is now unless
__host__ error_t Sampler_Grid::convertPreMapper(Vec3f *dir_list, int32_t size) {
	convert_pre_mapper <<< 1, block_w * block_h >>> (buffer_dir, dir_list, w, h, block_w, block_h, grid_w, grid_h);
	return ERROR_NO;
}


// TODO: current assume that size is known and size is now unless
__host__ error_t Sampler_Grid::convertPostMapper(Vec3f *image_list, int32_t size) {
	convert_post_mapper <<< 1, block_w * block_h >>> (buffer_image, image_list, w, h, block_w, block_h, grid_w, grid_h);
	return ERROR_NO;
}


__host__ void Sampler_Grid::update() {
	// check if w, h, block_w, block_h is valid or not
	// update operation will run without error iff these variables are valid
	if (w == 0 || h == 0 || block_w == 0 || block_h == 0) return;

	// deallocate previous space
	hipFree(buffer_dir);
	hipFree(buffer_image);

	// to ensure that each kernel block is in same size
	// padding is used for buffer
	// therefore, size of buffer may be larger than direction vector list and image list
	//
	// CURRENT: actual buffer size depends on grid size
	// CURRENT: i.e. setSizeGrid is needed to be called before this function
	int32_t	w_padded = w;
	int32_t	h_padded = h;
	grid_w	= w / block_w;
	grid_h	= h / block_h;

	if (w % block_w != 0) {
		w_padded = w + block_w;
		++grid_w;
	}
	if (h % block_h != 0) {
		h_padded = h + block_h;
		++grid_h;
	}
	
	// allocate (device) space for buffer and set size
	hipMalloc(&buffer_dir, w_padded * h_padded * sizeof(Vec3f));
	hipMalloc(&buffer_image, w_padded * h_padded * sizeof(Vec3f));

	buffer_dir_size		= w_padded * h_padded * sizeof(Vec3f);
	buffer_image_size	= w_padded * h_padded * sizeof(Vec3f);
}


// Static Function Implementation
__global__ void convert_pre_mapper(
	Vec3f *dst, Vec3f *src, int32_t src_w, int32_t src_h,
	int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h) {

	// global index
	int	global_index	= blockIdx.x * blockDim.x + threadIdx.x;
	int	global_x		= global_index % block_w;
	int	global_y		= global_index / block_w;

	// TODO: currently offset is equal to block_w * block_h
	// but this limit the size of thread
	int32_t offset = block_w * block_h;
	int32_t i = 0;

	// for every grid
	for (int32_t y = 0; y < grid_h; ++y) {
		for (int32_t x = 0; x < grid_w; ++x) {

			// global xy
			const int32_t src_x = x * block_w + global_x; 
			const int32_t src_y = y * block_h + global_y;

			if (src_x > src_w || src_y > src_h)	dst[i + global_index] = Vec3f(0);
			else								dst[i + global_index] = src[src_y * src_w + src_x];
		}

		i += offset;
	}
}


__global__ void convert_post_mapper(
	Vec3f *dst, Vec3f *src, int32_t dst_w, int32_t dst_h, 
	int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h) {

	// global index
	int	global_index	= blockIdx.x * blockDim.x + threadIdx.x;
	int	global_x		= global_index % block_w;
	int	global_y		= global_index / block_w;

	// TODO: currently offset is equal to block_w * block_h
	// but this limit the size of thread
	int32_t	offset = block_w * block_h;

	for (int32_t i = 0; i < grid_w * grid_h; ++i) {

		// grid xy
		const int32_t src_x = i % grid_w;
		const int32_t src_y = i / grid_w;

		// global xy
		const int32_t dst_x = src_x + global_x;
		const int32_t dst_y = src_y + global_y;

		if (dst_x < dst_w && dst_y < dst_y) {
			dst[dst_y * dst_w + dst_x] = src[i * offset + global_index];
		}

	}
}

#include "../inc/RayTracer_Sampler_Grid.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__host__ Sampler_Grid::Sampler_Grid():
grid_w(1),
grid_h(1)
{
}


__host__ Sampler_Grid::~Sampler_Grid()  {
}


__host__ error_t Sampler_Grid::setSizeGrid(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;

	// TODO: in this current condition, grid and block mean the same thing
	this->grid_w	= w;
	this->grid_h	= h;

	this->block_w	= w;
	this->block_h	= h;

	return ERROR_NO;
}


__host__ error_t Sampler_Grid::setSizeImage(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;
	
	// set image size
	Sampler::setSizeImage(w, h);

	// deallocate previous space
	// hipFree(buffer_dir);
	// hipFree(buffer_image);

	// to ensure that each kernel block is in same size
	// padding is used for buffer
	// therefore, size of buffer may be larger than direction vector list and image list
	//
	// CURRENT: actual buffer size depends on grid size
	// CURRENT: i.e. setSizeGrid is needed to be called before this function
	int32_t	w_padded = w;
	int32_t	h_padded = h;

	if (w % grid_w != 0) w_padded = w + grid_w;
	if (h % grid_h != 0) h_padded = h + grid_h;
	
	// allocate (device) space for buffer
	hipMalloc(&buffer_dir, w_padded * h_padded * sizeof(Vec3f));
	hipMalloc(&buffer_image, w_padded * h_padded * sizeof(Vec3f));

	return ERROR_NO;
}


// TODO: not yet completed
__host__ error_t Sampler_Grid::convertPreMapper(Vec3f *dir_list, int32_t size) {
	// from dir_list to buffer_dir
}


// TODO: not yet completed
__host__ error_t Sampler_Grid::convertPostMapper(Vec3f *image_list, int32_t size) {
	// from buffer_image to image_list
}


// Static Function Implementation
// ...

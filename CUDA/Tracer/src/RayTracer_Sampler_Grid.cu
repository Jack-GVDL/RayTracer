#include "hip/hip_runtime.h"
#include "../inc/RayTracer_Sampler_Grid.cuh"

// TODO: test
#include <stdio.h>


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
__global__ void	convert_pre_mapper	(fp_t *dst, fp_t *src, int32_t src_w, int32_t src_h,
									 int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h);
__global__ void	convert_post_mapper	(fp_t *dst, fp_t *src, int32_t dst_w, int32_t dst_h, 
									 int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h);


// Operation Handling
__host__ Sampler_Grid::Sampler_Grid():
grid_w(0),
grid_h(0)
{
}


__host__ Sampler_Grid::~Sampler_Grid()  {
}


__host__ error_t Sampler_Grid::setSizeGrid(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;

	this->block_w	= w;
	this->block_h	= h;

	update();
	return ERROR_NO;
}


// backup
/*
__host__ error_t Sampler_Grid::setSizeImage(int32_t w, int32_t h) {
	if (w <= 0 || h <= 0) return ERROR_ANY;
	
	// set image size
	Sampler::setSizeImage(w, h);

	update();
	return ERROR_NO;
}
*/


// TODO: current assume that size is known and size is now unless
__host__ error_t Sampler_Grid::convertPreMapper(fp_t *dir_list, int32_t size) {
	convert_pre_mapper <<< 1, block_w * block_h >>> (buffer_dir, dir_list, w, h, block_w, block_h, grid_w, grid_h);
	// hipDeviceSynchronize();

	// TODO: test
	// printf("convertPreMapper \n");

	return ERROR_NO;
}


// TODO: current assume that size is known and size is now unless
__host__ error_t Sampler_Grid::convertPostMapper(fp_t *image_list, int32_t size) {
	convert_post_mapper <<< 1, block_w * block_h >>> (image_list, buffer_image, w, h, block_w, block_h, grid_w, grid_h);
	// hipDeviceSynchronize();
	return ERROR_NO;
}


__host__ void Sampler_Grid::update() {
	// check if w, h, block_w, block_h is valid or not
	// update operation will run without error iff these variables are valid
	if (w == 0 || h == 0 || block_w == 0 || block_h == 0) return;

	// deallocate previous space
	hipFree(buffer_dir);
	hipFree(buffer_image);

	// to ensure that each kernel block is in same size
	// padding is used for buffer
	// therefore, size of buffer may be larger than direction vector list and image list
	//
	// CURRENT: actual buffer size depends on grid size
	// CURRENT: i.e. setSizeGrid is needed to be called before this function
	int32_t	w_padded = w;
	int32_t	h_padded = h;
	grid_w	= w / block_w;
	grid_h	= h / block_h;

	if (w % block_w != 0) ++grid_w;
	if (h % block_h != 0) ++grid_h;

	w_padded = grid_w * block_w;
	h_padded = grid_h * block_h;
	
	// allocate (device) space for buffer and set size
	buffer_dir_size		= w_padded * h_padded * 2 * sizeof(fp_t);
	buffer_image_size	= w_padded * h_padded * 3 * sizeof(fp_t);

	hipMalloc(&buffer_dir,		buffer_dir_size);
	hipMalloc(&buffer_image,	buffer_image_size);

	// set padded image size
	size_padded = w_padded * h_padded;
}


// Static Function Implementation
__global__ void convert_pre_mapper(
	fp_t *dst, fp_t *src, int32_t src_w, int32_t src_h,
	int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h) {

	// global index
	int	global_index	= blockIdx.x * blockDim.x + threadIdx.x;
	int	global_x		= global_index % block_w;
	int	global_y		= global_index / block_w;

	// TODO: test
	// printf("Pre mapper: %i %i, %i %i, %i %i \n", block_w, block_h, grid_w, grid_h, global_x, global_y);

	// TODO: currently offset is equal to block_w * block_h
	// but this limit the size of thread
	int32_t offset = block_w * block_h;
	int32_t i = 0;

	// for every grid
	for (int32_t y = 0; y < grid_h; ++y) {
		for (int32_t x = 0; x < grid_w; ++x) {

			// global xy
			const int32_t src_x = x * block_w + global_x; 
			const int32_t src_y = y * block_h + global_y;
			
			// TODO: test
			// if (global_index == 0) printf("%i %i %i %i \n", y, x, i, src_y * src_w + src_x);

			if (src_x > src_w || src_y > src_h)	{
				dst[(i + global_index) * 2 + 0] = 0.0;
				dst[(i + global_index) * 2 + 1] = 0.0;
			} else {
				dst[(i + global_index) * 2 + 0] = src[(src_y * src_w + src_x) * 2 + 0];
				dst[(i + global_index) * 2 + 1] = src[(src_y * src_w + src_x) * 2 + 1];
			}
			
			i += offset;
		}
	}
}


__global__ void convert_post_mapper(
	fp_t *dst, fp_t *src, int32_t dst_w, int32_t dst_h, 
	int32_t block_w, int32_t block_h, int32_t grid_w, int32_t grid_h) {

	// global index
	int	global_index	= blockIdx.x * blockDim.x + threadIdx.x;
	int	global_x		= global_index % block_w;
	int	global_y		= global_index / block_w;

	// TODO: currently offset is equal to block_w * block_h
	// but this limit the size of thread
	int32_t	offset = block_w * block_h;

	for (int32_t i = 0; i < grid_w * grid_h; ++i) {

		// grid xy
		const int32_t src_x = i % grid_w;
		const int32_t src_y = i / grid_w;

		// global xy
		const int32_t dst_x = src_x * block_w + global_x;
		const int32_t dst_y = src_y * block_h + global_y;

		if (dst_x < dst_w && dst_y < dst_h) {
			dst[(dst_y * dst_w + dst_x) * 3 + 0] = src[(i * offset + global_index) * 3 + 0];
			dst[(dst_y * dst_w + dst_x) * 3 + 1] = src[(i * offset + global_index) * 3 + 1];
			dst[(dst_y * dst_w + dst_x) * 3 + 2] = src[(i * offset + global_index) * 3 + 2];
		}

	}

	// TODO: test
	// printf("Enter \n");
}

#include "../inc/RayTracer_Tracer.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ RayTracer::RayTracer() {
	// TODO: temporary allocate some space for memory control
	// void *memory = malloc(sizeof(uint8_t) * 1024 * 50);
	void *memory;
	hipMalloc(&memory, 1024 * 50 * sizeof(uint8_t));

	scheduler.memory_control.setMemory(memory, sizeof(uint8_t) * 1024 * 20);
}


__device__ RayTracer::~RayTracer() {
}


__device__ void RayTracer::setScene(Scene *scene) {
	this->scene = scene;
	scheduler.setScene(scene);
}


__device__ Vec3f RayTracer::trace(const Camera *camera, fp_t x, fp_t y, int32_t depth) {
	Ray ray = camera->getRay(x, y);
	return trace(&ray, depth);
}


__device__ Vec3f RayTracer::trace(const Ray *ray, int depth) {
	// init scatter record
	RecordRay scatter_record;
	scatter_record.parent	= nullptr;
	scatter_record.scene	= scene;
	scatter_record.depth	= depth;

	RecordHit *record_hit	= &(scatter_record.record_hit.record);
	record_hit->ray 		= *ray;

	return trace(&scatter_record);
}


__device__ Vec3f RayTracer::trace(RecordRay *record) {
	// set record
	scheduler.setRoot(record);

	// run the scheduler until there is nothing to be scheduled
	while (scheduler.schedule());

	// return intensity
	scheduler.getRoot(record);
	return record->intensity;
}


// Static Function Implementation
// ...

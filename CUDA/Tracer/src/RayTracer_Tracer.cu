#include "../inc/RayTracer_Tracer.cuh"


// Define
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__ Tracer::Tracer() {
	// TODO: remove
	// TODO: temporary allocate some space for memory control
	// void *memory = malloc(sizeof(uint8_t) * 1024 * 50);
	// void *memory;
	// hipMalloc(&memory, 1024 * 50 * sizeof(uint8_t));
	// scheduler.memory_control.setMemory(memory, sizeof(uint8_t) * 1024 * 20);
}


__device__ Tracer::~Tracer() {
}


__device__ void Tracer::setScene(Scene *scene) {
	this->scene = scene;

	// TODO: remove
	// scheduler.setScene(scene);
}


__device__ error_t Tracer::addScheduler(Scheduler_Scatter *scheduler) {
	if (scheduler == nullptr) return ERROR_ANY;

	scheduler_list.push_back(&scheduler, sizeof(Scheduler_Scatter*));
	return ERROR_NO;
}


// TODO: not yet completed
__device__ error_t Tracer::rmScheduler(Scheduler_Scatter *scheduler) {
	return ERROR_ANY;
}


// backup
/*
__device__ void Tracer::resetScheduler() {
	scheduler_index = 0;
}
*/


__device__ Vec3f Tracer::trace(const Camera *camera, fp_t x, fp_t y, int32_t depth, int32_t index) {
	Ray ray = camera->getRay(x, y);
	return trace(&ray, depth, index);
}


__device__ Vec3f Tracer::trace(const Ray *ray, int depth, int32_t index) {
	// init scatter record
	RecordRay scatter_record;
	scatter_record.parent	= nullptr;
	scatter_record.scene	= scene;
	scatter_record.depth	= depth;

	RecordHit *record_hit	= &(scatter_record.record_hit.record);
	record_hit->ray 		= *ray;

	return trace(&scatter_record, index);
}


__device__ Vec3f Tracer::trace(RecordRay *record, int32_t index) {
	// request for a scheduler
	int32_t scheduler_size = scheduler_list.index / sizeof(Scheduler_Scatter*);
	if (index >= scheduler_size) return Vec3f();

	Scheduler_Scatter *scheduler = ((Scheduler_Scatter**)scheduler_list.list)[index];

	// first set record
	// then run the scheduler until there is nothing to be scheduled
	scheduler->setScene(scene);
	scheduler->setRoot(record);

	while (scheduler->schedule());

	// return intensity
	scheduler->getRoot(record);
	return record->intensity;
}


// Static Function Implementation
// ...

#include "../inc/RayTracer_UtilMath.cuh"
#include "../inc/Util_Random.cuh"


// Define
// using namespace UtilMath;


// Typedef
// ...


// Static Data
__device__ Random	*random;
__device__ fp_t		*table_random;
__device__ int32_t	size_table;
__device__ int32_t	offset = 0;


// Static Function Prototype
// ...


// Operation Handling
namespace UtilMath {

__device__ void initUtilMath(int32_t size_random) {
	random = new Random();

	hipMalloc(&table_random, size_random * sizeof(fp_t));
	size_table = size_random;

	for (int32_t i = 0; i < size_table; ++i) table_random[i] = randFloat();
}


__device__ fp_t randFloat() {
	return random->randUniform();
}

__device__ fp_t randFloat(int32_t index) {
	++offset;
	return table_random[(index + offset) % size_table];
}

}


// Static Function Implementation
// ...

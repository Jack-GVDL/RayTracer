
#include "../inc/Util_Random.cuh"


// Define
// ...


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
__device__	Random::Random() {
	hiprand_init(0, 0, 0, &state);
}


__device__	Random::~Random() {

}

__device__ fp_t Random::randUniform() {
	return hiprand_uniform(&state);
}


// Static Function Implementation
// ...

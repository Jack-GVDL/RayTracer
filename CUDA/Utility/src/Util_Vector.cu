#include "../inc/Util_Vector.cuh"


// Define
// TODO: this value should not be in define region
#define FACTOR_EXTEND	2


// Typedef
// ...


// Static Data
// ...


// Static Function Prototype
// ...


// Operation Handling
// __init__
__device__ Vector::Vector() {
	hipMalloc(&list, size);
}


__device__ Vector::Vector(int32_t size) {
	this->base	= size;
	this->size	= base;
	hipMalloc(&list, size);
}


__device__ Vector::~Vector() {
}


// operation
__device__ void Vector::push_back(void *data, int32_t s) {
	// check if need to push
	if (index + s > size) {

		// find the size need to extend
		int size_new = size;
		while (size_new < index + s) size_new *= FACTOR_EXTEND;

		// allocate new space
		uint8_t *list_new;
		hipMalloc(&list_new, size_new);

		// copy to new space
		// hipMemcpyAsync(list_new, list, index, hipMemcpyDeviceToDevice);
		memcpy(list_new, list, index);

		// update list info
		size = size_new;
		list = list_new;
	}

	// add data to the list
	// copy to the list
	// hipMemcpyAsync(list + index, data, s, hipMemcpyDeviceToDevice);
	memcpy(list + index, data, s);
	index += s;
}


// TODO: not yet completed
/*
__device__ void Vector::pop_back(int32_t s) {
}


__device__ void Vector::front() {
}


__device__ void Vector::back() {
}
*/


// Static Function Implementation
// ...

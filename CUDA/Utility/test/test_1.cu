#include "hip/hip_runtime.h"
// This test file is to test if Vec3f can be function properly


#include <stdio.h>
#include "../Utility.cuh"


// device
__global__ void increment_1(Vec3f *list, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) list[i] += 1;
}


// host
int main() {
	// data
	printf("Stage: Data \n");
	const int size = 1000;

	Vec3f *vec_list_host;
	Vec3f *vec_list_device;

	vec_list_host = (Vec3f*)malloc(size * sizeof(Vec3f));
	hipMalloc(&vec_list_device, size * sizeof(Vec3f));

	for (int i = 0; i < size; ++i) vec_list_host[i] = Vec3f(i, i + 1, i + 2);
	hipMemcpy(vec_list_device, vec_list_host, size * sizeof(Vec3f), hipMemcpyHostToDevice);

	// kernel
	printf("Stage: Kernel \n");
	increment_1<<< (size + 255) / 256, 256 >>>(vec_list_device, size);

	// copy back
	printf("Stage: Copy back \n");
	hipMemcpy(vec_list_host, vec_list_device, size * sizeof(Vec3f), hipMemcpyDeviceToHost);

	// result
	printf("Stage: Result \n");
	for (int i = 0; i < size; ++i) {
		if (vec_list_host[i][0] == i + 0 + 1 &&
			vec_list_host[i][1] == i + 1 + 1 &&
			vec_list_host[i][2] == i + 2 + 1) continue;

		printf("Error \n");
		break;
	}
	
	return 0;
}
